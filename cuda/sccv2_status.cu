#include "hip/hip_runtime.h"
#include "../utils/is_checked.cu"
#include "../utils/file2graph_notunsigned.cpp"
#include "scc_operations.cu"
#include <cstring>
#include <hip/hip_runtime.h>
using namespace std;

#define DEBUG_F_KERNEL false
#define DEBUG_REACH false
#define DEBUG_TRIMMING_KERNEL false
#define DEBUG_TRIMMING false
#define DEBUG_UPDATE false
#define DEBUG_FW_BW false
#define DEBUG_MAIN false
#define DEBUG_FINAL true

/*

VERSIONE DEL CODICE CUDA: NAIVEv2

Rispetto alla versione precedente, viene utilizzato al posto dei vettori booleani, il vettore "status". Ogni i-esimo elemento occupa un byte: 6 degli 8 bit che compongono
tale byte rappresentano lo stato di uno dei vettori booleani

*/

__global__ void f_kernel(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, char * d_status, bool * d_stop, bool (*get_visited)(char *), bool (*get_expanded)(char *), void (*set_visited)(char *), void (*set_expanded)(char *)){
	// Esecuzione di un thread della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

    // Per ogni nodo
	if(v < num_nodes) {
        // Si controlla se non è stato eliminato E è stato eliminato E non è stato espanso
		if(!get_is_d_eliminated(&d_status[v]) && get_visited(&d_status[v]) && !get_expanded(&d_status[v])) {
            // Si segna come espanso
			set_expanded(&d_status[v]);

            // Per ogni nodo a cui punta
			for(int u = d_nodes[v]; u < d_nodes[v + 1]; u++) {	
				int dst = d_adjacency_list[u];

                // Si controlla se non è stato eliminato E se non è stato visitato E se il colore del nodo che punta corrisponde a quello del nodo puntato
				if(!get_is_d_eliminated(&d_status[dst]) && !get_visited(&d_status[dst]) && d_pivots[v] == d_pivots[dst]) {
                    // Setta il nodo puntato a visitato
					set_visited(&d_status[dst]);
                    // Permette di continuare il ciclo in reach, perchè si è trovato un altro nodo da visitare
					*d_stop = false;
				}
			}
		}
	}
}

void reach(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, char * d_status, bool (*get_visited)(char *), bool (*get_expanded)(char *), void (*set_visited)(char *), void (*set_expanded)(char *), const int n_blocks, const int t_per_blocks) {
	// Esecuzione ricorsiva della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming

	bool stop, *d_stop;
	stop = false;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));
	
    // Si effettua la chiusura in avanti/indietro
    while(!stop) {
		HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
        f_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, d_stop, get_visited, get_expanded, set_visited, set_expanded);	
		HANDLE_ERROR(hipMemcpy(&stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
    }
	
	HANDLE_ERROR(hipFree(d_stop));
}

__global__ void trimming_kernel(int num_nodes, int * d_nodes, int * d_nodes_transpose, int * d_adjacency_list,  int * d_adjacency_list_transpose, char * d_status, bool * d_stop){
	// Esegue un'eliminazione di nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes) {
		if(!get_is_d_eliminated(&d_status[v])){
			// Se questo valore non verrà cambiato, allora il nodo verrà cancellato
			bool elim = true;

			bool forward = false;
			bool backward = false;
			
			// Nel caso un nodo abbia entrambi in_degree o out_degree diversi da 0, tra i soli nodi non eliminati, allora non va eliminato
			for(int u = d_nodes[v]; u < d_nodes[v+1]; u++){
				if(!get_is_d_eliminated(&d_status[d_adjacency_list[u]])) {
					forward = true;
				}
			}
			if(forward) {
				for(int u = d_nodes_transpose[v]; u < d_nodes_transpose[v+1]; u++){
					if(!get_is_d_eliminated(&d_status[d_adjacency_list_transpose[u]])) {
						backward = true;
					}
				}
			}
			if(backward) {
				elim = false;
			}

			if(elim){
				set_is_d_eliminated(&d_status[v]);
				*d_stop = false;
			}
		}
	}
}

void trimming(int num_nodes, int * d_nodes, int * d_nodes_transpose, int * d_adjacency_list, int * d_adjacency_list_transpose, char * d_status, const int n_blocks, const int t_per_blocks) {
	// Elimina iterativamente i nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming

	bool stop, *d_stop;
	stop = false;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));

    while(!stop) {
		HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
        trimming_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, d_stop);
		HANDLE_ERROR(hipMemcpy(&stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
    }

	HANDLE_ERROR(hipFree(d_stop));
}

__global__ void set_colors(int num_nodes, char * d_status, int * d_pivots, int * d_colors, long * d_write_id_for_pivots, bool * d_stop){
	// Esegue l'update dei valori del pivot facendo una race, scrivendo il "colore" di una serie di pivot in array simultaneamente
	// @param:	pivots						= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated				= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited				= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited				= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: d_write_id_for_pivots		= Lista che conterrà, nelle posizione identificate dai colori appena calcolati, i nuovi pivot da assegnare
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes) {
		if(get_is_d_eliminated(&d_status[v])){
			d_pivots[v] = v;
		}

		const bool fw_visitated = get_is_d_fw_visited(&d_status[v]);
		const bool bw_visitated = get_is_d_bw_visited(&d_status[v]);
		
		if(fw_visitated == bw_visitated && fw_visitated == true){
			d_colors[v] = 4 * d_pivots[v];
		} else {
			if(fw_visitated != bw_visitated && fw_visitated == true){
				d_colors[v] = 4 * d_pivots[v] + 1;
			}else if(fw_visitated != bw_visitated && fw_visitated == false){
				d_colors[v] = 4 * d_pivots[v] + 2;
			}else if(fw_visitated == bw_visitated && fw_visitated == false){
				d_colors[v] = 4 * d_pivots[v] + 3;				
			}
				
			if(!get_is_d_eliminated(&d_status[v])){
				*d_stop = false;
			}
		}
		d_write_id_for_pivots[d_colors[v]] = v;
	}
}

__global__ void set_race_winners(int num_nodes, char * d_status, int * d_pivots, int * d_colors, long * d_write_id_for_pivots){
	// Ottenuti i valori della race, si vanno ad impostare i nuovi pivot
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update

	
	int v = threadIdx.x + blockIdx.x * blockDim.x;
	if(v < num_nodes) {
		// Se il nodo è stato eliminato, allora il suo pivot è per forza se stesso
		if(get_is_d_eliminated(&d_status[v])){
			d_pivots[v] = v;
		}else{
			d_pivots[v] = d_write_id_for_pivots[d_colors[v]];
			set_is_d_fw_visited(&d_status[d_pivots[v]]);
			set_is_d_bw_visited(&d_status[d_pivots[v]]);
		}
	}
}

__global__ void initialize_pivot(int num_nodes, int * d_pivots, char * d_status) {
	// Scelta iniziale del primo pivot, basandosi sui nodi cancellati inizialmente
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, avente come pivot un nodo non cancellato
	//          fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no. A questo punto l'unico nodo visitato è il solo pivot scelto
	//          bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no. A questo punto l'unico nodo visitato è il solo pivot scelto

	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes){
		__shared__ int chosen_pivot;
		if(!get_is_d_eliminated(&d_status[v])){
			chosen_pivot = v;
		}

		// Sincronizziamo qui i thread per inizializzare questi array: lanciare un altro thread
		// solo per inizializzare gli array potrebbe risultare più pesante che farlo qui
		__syncthreads();

		d_pivots[v] = chosen_pivot;
		set_is_d_fw_visited(&d_status[d_pivots[v]]);
		set_is_d_bw_visited(&d_status[d_pivots[v]]);
	}
}

void update(int num_nodes, int * d_pivots, char * d_status, long * d_write_id_for_pivots, int * d_colors, bool * stop, const int n_blocks, const int t_per_blocks) {
	// Esegue l'update dei valori del pivot facendo una race
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update


	bool *d_stop;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));
	
	HANDLE_ERROR(hipMemset(d_write_id_for_pivots, -1, 4 * num_nodes * sizeof(long)));
	HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
	
	// Dai paper:
	// These subgraphs are 
	// 		1) the strongly connected component with the pivot;
	// 		2) the subgraph given by vertices in the forward closure but not in the backward closure; 
	// 		3) the subgraph given by vertices in the backward closure but not in the forward closure;
	// 		4) the subgraph given by vertices that are neither in the forward nor in the backward closure.
	
	// The subgraphs that do not contain the pivot form three independent instances of the same problem, and therefore, 
	// they are recursively processed in parallel with the same algorithm
	
	set_colors<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots, d_stop);
	
	HANDLE_ERROR(hipMemcpy(stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(d_stop));

	// Setto i valori dei pivot che hanno vinto la race
	// Se sono stati eliminati, allora setta il valore dello stesso nodo 
	set_race_winners<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots);
}

__global__ void trim_u_kernel(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, char * d_status, int * d_is_scc){
	// Setta i pivot delle SCC uguale a -1 se questi ricevono archi da nodi u
	// param: 	pivots = 	Lista che per ogni 'v' dice il valore del pivot della SCC
	// 			is_scc =	Lista copia di pivots
	// @return:	is_scc =	Lista contenente i pivot delle SCC, però i pivot delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes){
		if(get_is_d_u(&d_status[v])){
			for(int u = d_nodes[v]; u < d_nodes[v+1]; ++u) {
				if(d_pivots[v] != d_pivots[d_adjacency_list[u]]) {
					d_is_scc[d_pivots[d_adjacency_list[u]]] = -1;
				}
			}
		}
	}

}


void routine_v2(const bool profiling, int num_nodes, int num_edges, int * nodes, int * adjacency_list, int * nodes_transpose, int * adjacency_list_transpose, char * status) {
	// Impostazione del device
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

    int * is_scc;

	const int THREADS_PER_BLOCK = prop.maxThreadsPerBlock;
	const int NUMBER_OF_BLOCKS = num_nodes / THREADS_PER_BLOCK + (num_nodes % THREADS_PER_BLOCK == 0 ? 0 : 1);

	// Dichiarazioni di variabili device
	int * d_is_scc, * d_more_than_one, * d_nodes, * d_adjacency_list, * d_nodes_transpose, * d_adjacency_list_transpose, * d_pivots, * d_colors;
	char * d_status;
	long * d_write_id_for_pivots;

	HANDLE_ERROR(hipMalloc((void**)&d_nodes, (num_nodes+1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_nodes_transpose, (num_nodes+1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_adjacency_list, num_edges * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_adjacency_list_transpose, num_edges * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_pivots, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_colors, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_status, num_nodes * sizeof(char)));

	HANDLE_ERROR(hipMalloc((void**)&d_write_id_for_pivots, 4 * num_nodes * sizeof(long)));

	// Le strutture principali le copiamo nel device già qui, visto che non verranno mai modificate
	HANDLE_ERROR(hipMemcpy(d_nodes, nodes, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_adjacency_list, adjacency_list, num_edges * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_nodes_transpose, nodes_transpose, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_adjacency_list_transpose, adjacency_list_transpose, num_edges * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(d_status, status, num_nodes * sizeof(char), hipMemcpyHostToDevice));

	// Inizializzazione e copia delle funzioni device che verranno passate tramite parametro.
	// Utilizzando le funzioni in questo modo, anche se apparentemente verboso, permette di ottenere meno codice duplicato:
	// infatti, se non fosse per queste variabili, si sarebbe dovuto duplicare l'f_kernel e il reach per averne uno per la forward e uno per la backward.
	get_status h_get_fw_visited, h_get_bw_visited, h_get_fw_expanded, h_get_bw_expanded;
	set_status h_set_fw_visited, h_set_bw_visited, h_set_fw_expanded, h_set_bw_expanded;

	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_fw_visited, HIP_SYMBOL(dev_get_fw_visited), sizeof(get_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_bw_visited, HIP_SYMBOL(dev_get_bw_visited), sizeof(get_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_fw_expanded, HIP_SYMBOL(dev_get_fw_expanded), sizeof(get_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_bw_expanded, HIP_SYMBOL(dev_get_bw_expanded), sizeof(get_status)));

	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_fw_visited, HIP_SYMBOL(dev_set_fw_visited), sizeof(set_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_bw_visited, HIP_SYMBOL(dev_set_bw_visited), sizeof(set_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_fw_expanded, HIP_SYMBOL(dev_set_fw_expanded), sizeof(set_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_bw_expanded, HIP_SYMBOL(dev_set_fw_expanded), sizeof(set_status)));
	
	// Primo trimming per eliminare i nodi che, dopo la cancellazione dei nodi non in U,
	// non avevano più out-degree e in-degree diverso da 0
	trimming(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, THREADS_PER_BLOCK, NUMBER_OF_BLOCKS);
	
	// Si fanno competere i thread per scelgliere un nodo che farà da pivot, a patto che quest'ultimo sia non eliminato
	initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);
	
    bool stop = false;
	
	// Si ripete il ciclo fino a quando tutti i nodi vengono eliminati
    while (!stop){
		// Forward reach
		DEBUG_MSG("Forward reach:" , "", DEBUG_FW_BW);
        reach(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, h_get_fw_visited, h_get_fw_expanded, h_set_fw_visited, h_set_fw_expanded, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
		
		// Backward reach
        DEBUG_MSG("Backward reach:" , "", DEBUG_FW_BW);
		reach(num_nodes, d_nodes_transpose, d_adjacency_list_transpose, d_pivots, d_status, h_get_bw_visited, h_get_bw_expanded, h_set_bw_visited, h_set_bw_expanded, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		// Trimming per eliminare ulteriori nodi che non hanno più out-degree e in-degree diversi da 0
		DEBUG_MSG("Trimming:" , "", DEBUG_FW_BW);
        trimming(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		// Update dei pivot
		DEBUG_MSG("Update:" , "", DEBUG_FW_BW);
		update(num_nodes, d_pivots, d_status, d_write_id_for_pivots, d_colors, &stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
    }
	
	//Disallocamento della memoria iniziale
	HANDLE_ERROR(hipFree(d_write_id_for_pivots));
	
	// Tramite fw_bw_ abbiamo ottenuto, per ogni nodo, il pivot della SCC a cui appartiene.
	// Allochiamo is_scc, che alla fine avrà per ogni nodo il pivot della sua SCC se la sua SCC è accettabile, altrimenti -1
	
	// Per iniziare le assegnamo gli stessi valori di pivots, che verranno modificati in seguito
	is_scc = (int*) malloc(num_nodes * sizeof(int));
	
	// Allochiamo more_than_one, che per ogni nodo che fa da pivot viene assegnato un contatore, il quale conta quante volte appare tale pivot
	// Se appare solo una volta, allora il nodo non fa parte di nessuna SCC
	HANDLE_ERROR(hipMalloc((void**)&d_is_scc, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_more_than_one, num_nodes * sizeof(int)));
	
	HANDLE_ERROR(hipMemcpy(d_is_scc, d_pivots, num_nodes * sizeof(int), hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemset(d_more_than_one, 0, num_nodes * sizeof(int)));

	trim_u_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, d_is_scc);
	
	HANDLE_ERROR(hipFree(d_adjacency_list_transpose));
	HANDLE_ERROR(hipFree(d_adjacency_list));
	HANDLE_ERROR(hipFree(d_nodes_transpose));
	HANDLE_ERROR(hipFree(d_nodes));
	
	trim_u_propagation_v1<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_is_scc);

	if(profiling){
		bool * d_is_scc_final;
		HANDLE_ERROR(hipMalloc((void**)&d_is_scc_final, num_nodes * sizeof(bool)));
		convert_int_array_to_bool<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_is_scc, d_is_scc_final);
		eliminate_trivial_scc<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, THREADS_PER_BLOCK*sizeof(unsigned int) + THREADS_PER_BLOCK*sizeof(bool)>>>(THREADS_PER_BLOCK, num_nodes, (unsigned int*)d_pivots, d_is_scc_final);
		hipDeviceSynchronize();
		
		bool result = or_reduce(THREADS_PER_BLOCK, num_nodes, d_is_scc_final);
		printf("%d", result);
		HANDLE_ERROR(hipFree(d_is_scc_final));
	}else{
		calculate_more_than_one<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_more_than_one, d_is_scc);
		is_scc_adjust_v1<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_more_than_one, d_is_scc);
		HANDLE_ERROR(hipMemcpy(is_scc, d_is_scc, num_nodes * sizeof(int), hipMemcpyDeviceToHost));
		DEBUG_MSG("Number of SCCs found: ", count_distinct_scc_v1(is_scc, num_nodes), DEBUG_FINAL);
	}

	HANDLE_ERROR(hipFree(d_pivots));
	HANDLE_ERROR(hipFree(d_more_than_one));
	HANDLE_ERROR(hipFree(d_status));
	HANDLE_ERROR(hipFree(d_is_scc));

}