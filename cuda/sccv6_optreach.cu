#include "../utils/is_checked.cu"
#include "../utils/file2graph.cpp"
#include "scc_operations.cu"
#include <cstring>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <set>
using namespace std;

#define DEBUG_F_KERNEL false
#define DEBUG_REACH false
#define DEBUG_TRIMMING_KERNEL false
#define DEBUG_TRIMMING false
#define DEBUG_UPDATE false
#define DEBUG_FW_BW false
#define DEBUG_MAIN false
#define DEBUG_FINAL true

#define CUDA_STREAMS 9
#define OMP_MIN_NODES 100000
/*
VERSIONE DEL CODICE CUDA: SCCv5 - OpenMP
Rispetto alla quarta versione, in questa vengono parallelizzati, tramite le direttive di OpenMP, le chiamate all'API di CUDA per l'allocazione e il trasferimento dei dati
*/

void trimming_v6(unsigned int const num_nodes, unsigned int * d_nodes, unsigned int * d_nodes_transpose, unsigned int * d_adjacency_list, unsigned int * d_adjacency_list_transpose, char * d_status, bool * stop, bool * d_stop, const unsigned int n_blocks, const unsigned int t_per_blocks) {
	// Elimina iterativamente i nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming

	*stop = false;
    while(!*stop) {
		*stop = true;
        trimming_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, d_stop);
		// Dobbiamo aspettare che la copia venga effettuata anche se è mappata
		hipDeviceSynchronize();
    }
}

void update_v6(unsigned int const num_nodes, unsigned int * d_pivots, char * d_status, unsigned int * d_colors, unsigned long * d_write_id_for_pivots, bool * stop, bool * d_stop, const unsigned int n_blocks, const unsigned int t_per_blocks) {
	// Esegue l'update dei valori del pivot facendo una race
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update
	
	*d_stop = true;

	// Dai paper:
	// These subgraphs are 
	// 		1) the strongly connected component with the pivot;
	// 		2) the subgraph given by vertices in the forward closure but not in the backward closure; 
	// 		3) the subgraph given by vertices in the backward closure but not in the forward closure;
	// 		4) the subgraph given by vertices that are neither in the forward nor in the backward closure.
	// The subgraphs that do not contain the pivot form three independent instances of the same problem, and therefore, 
	// they are recursively processed in parallel with the same algorithm
	
	set_colors<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots, d_stop);
	hipDeviceSynchronize();

	// Setto i valori dei pivot che hanno vinto la race
	// Se sono stati eliminati, allora setta il valore dello stesso nodo 
	set_new_pivots<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots);
}

void routine_v6(const bool profiling, unsigned int num_nodes, unsigned int num_edges, unsigned * nodes, unsigned * adjacency_list, unsigned * nodes_transpose, unsigned * adjacency_list_transpose, char * status) {
	// Impostazione del device
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipSetDeviceFlags(hipDeviceMapHost);

	const short MAX_THREADS_OMP = omp_get_max_threads();

	bool * stop, * d_stop, * bw_stop, * d_bw_stop;

	// Dichiarazioni di variabili device
	unsigned int * d_nodes, * d_adjacency_list, * d_nodes_transpose, * d_adjacency_list_transpose, * d_pivots, * d_colors;
	char * d_status, * d_bw_status;
	unsigned long * d_write_id_for_pivots;

	// Page-locking delle strutture dati principali
	#pragma omp parallel sections if(num_nodes>OMP_MIN_NODES) num_threads(MAX_THREADS_OMP)
	{
		#pragma omp section 
		{
			HANDLE_ERROR(hipHostRegister(nodes, (num_nodes+1) * sizeof(unsigned int), hipHostRegisterDefault));
		}

		#pragma omp section 
		{
			HANDLE_ERROR(hipHostRegister(adjacency_list, num_edges * sizeof(unsigned int), hipHostRegisterDefault));
		}

		#pragma omp section 
		{
			HANDLE_ERROR(hipHostRegister(nodes_transpose, (num_nodes+1) * sizeof(unsigned int), hipHostRegisterDefault));
		}

		#pragma omp section 
		{
			HANDLE_ERROR(hipHostRegister(adjacency_list_transpose, num_edges * sizeof(unsigned int), hipHostRegisterDefault));
		}

		#pragma omp section 
		{
			HANDLE_ERROR(hipHostRegister(status, (num_nodes+1) * sizeof(char), hipHostRegisterDefault));
		}
	}

	const unsigned int THREADS_PER_BLOCK = prop.maxThreadsPerBlock;
	const unsigned int NUMBER_OF_BLOCKS = (num_nodes / THREADS_PER_BLOCK) + (num_nodes % THREADS_PER_BLOCK == 0 ? 0 : 1);

	// Inizializzazione e copia delle funzioni device che verranno passate tramite parametro.
	// Utilizzando le funzioni in questo modo, anche se apparentemente verboso, permette di ottenere meno codice duplicato:
	// infatti, se non fosse per queste variabili, si sarebbe dovuto duplicare l'f_kernel e il reach per averne uno per la forward e uno per la backward.
	get_status h_get_fw_visited, h_get_bw_visited, h_get_fw_expanded, h_get_bw_expanded;
	set_status h_set_fw_visited, h_set_bw_visited, h_set_fw_expanded, h_set_bw_expanded;

	hipStream_t stream[CUDA_STREAMS];

	// Creazione delle stream, allocazione delle variabili device e copia dei dati
	// La parallelizzazione della copia dei dati non è effettiva, in quanto il canale di comunicazione è solo uno
	// e quindi la copia dei dati è serializzata.
	#pragma omp parallel if(num_nodes>OMP_MIN_NODES) num_threads(MAX_THREADS_OMP)
	{
		#pragma omp for schedule(static) 
		for (short i = 0; i < CUDA_STREAMS; i++) {
			hipStreamCreate(&stream[i]);
		}

		#pragma omp barrier

		#pragma omp sections nowait
		{
			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_fw_visited, HIP_SYMBOL(dev_get_fw_visited), sizeof(get_status), 0, hipMemcpyDefault, stream[0]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_bw_visited, HIP_SYMBOL(dev_get_bw_visited), sizeof(get_status), 0, hipMemcpyDefault, stream[1]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_fw_expanded, HIP_SYMBOL(dev_set_fw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[2]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_fw_expanded, HIP_SYMBOL(dev_get_fw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[3]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_bw_expanded, HIP_SYMBOL(dev_get_bw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[4]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_fw_visited, HIP_SYMBOL(dev_set_fw_visited), sizeof(set_status), 0, hipMemcpyDefault, stream[5]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_bw_visited, HIP_SYMBOL(dev_set_bw_visited), sizeof(set_status), 0, hipMemcpyDefault, stream[6]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_bw_expanded, HIP_SYMBOL(dev_set_bw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[7]));
			}	
		}

		//Allocazione delle variabili device
		#pragma omp sections nowait
		{
			#pragma omp section 
			{
				HANDLE_ERROR(hipMallocAsync((void**)&d_write_id_for_pivots, 4 * num_nodes * sizeof(unsigned long), stream[0]));
				HANDLE_ERROR(hipMallocAsync((void**)&d_pivots, num_nodes * sizeof(unsigned int), stream[1]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMallocAsync((void**)&d_adjacency_list, num_edges * sizeof(unsigned int), stream[2]));
				HANDLE_ERROR(hipMallocAsync((void**)&d_adjacency_list_transpose, num_edges * sizeof(unsigned int), stream[3]));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipMallocAsync((void**)&d_nodes, (num_nodes+1) * sizeof(unsigned int), stream[4]));
				HANDLE_ERROR(hipMallocAsync((void**)&d_nodes_transpose, (num_nodes+1) * sizeof(unsigned int), stream[5]));
			}
			
			#pragma omp section 
			{
				HANDLE_ERROR(hipMallocAsync((void**)&d_status, (num_nodes+1) * sizeof(char), stream[6]));
				HANDLE_ERROR(hipMallocAsync((void**)&d_bw_status, (num_nodes+1) * sizeof(char), stream[7]));
				HANDLE_ERROR(hipMallocAsync((void**)&d_colors, num_nodes * sizeof(unsigned int), stream[8]));
			}
		}

		#pragma omp barrier
		
		// Sincronizzazione delle stream
		#pragma omp for schedule(static)
		for (short i = 2; i < 7; i++) {
			hipStreamSynchronize(stream[i]);
		}

		#pragma omp barrier

		// hipMemcpy per archi e nodi
		#pragma omp sections
		{
			#pragma omp section
			{
				HANDLE_ERROR(hipMemcpyAsync(d_adjacency_list, adjacency_list, num_edges * sizeof(unsigned int), hipMemcpyHostToDevice, stream[2]));				
			}

			#pragma omp section 
			{	
				HANDLE_ERROR(hipMemcpyAsync(d_adjacency_list_transpose, adjacency_list_transpose, num_edges * sizeof(unsigned int), hipMemcpyHostToDevice, stream[3]));
			}

			#pragma omp section
			{
				HANDLE_ERROR(hipMemcpyAsync(d_nodes, nodes, (num_nodes+1) * sizeof(unsigned int), hipMemcpyHostToDevice, stream[4]));				
			}

			#pragma omp section
			{
				HANDLE_ERROR(hipMemcpyAsync(d_nodes_transpose, nodes_transpose, (num_nodes+1) * sizeof(unsigned int), hipMemcpyHostToDevice, stream[5]));	
			} 

			#pragma omp section
			{
				HANDLE_ERROR(hipMemcpyAsync(d_status, status, (num_nodes+1) * sizeof(char), hipMemcpyHostToDevice, stream[6]));				
			}
		}	
	}

	HANDLE_ERROR(hipHostAlloc(&stop, sizeof(bool), hipHostMallocMapped));
	HANDLE_ERROR(hipHostGetDevicePointer(&d_stop, stop, 0));

	HANDLE_ERROR(hipHostAlloc(&bw_stop, sizeof(bool), hipHostMallocMapped));
	HANDLE_ERROR(hipHostGetDevicePointer(&d_bw_stop, bw_stop, 0));
	
	// Primo trimming per eliminare i nodi che, dopo la cancellazione dei nodi non in U,
	// non avevano più out-degree e in-degree diverso da 0
	trimming_v6(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

	/* Inizializzazione delle variabili di test per il controllo della correttezza dell'algoritmo
	char * status_tmp, * bw_status_tmp;
	unsigned int * pivots_tmp;
	pivots_tmp = (unsigned int *) malloc(num_nodes * sizeof(unsigned int));
	status_tmp = (char *) malloc(num_nodes * sizeof(char));
	bw_status_tmp = (char *) malloc(num_nodes * sizeof(char)); */

	// Sincronizzazione implicita perché si utilizza il default stream
	// Si fanno competere i thread per scelgliere un nodo che farà da pivot, a patto che quest'ultimo sia non eliminato
	initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);
	hipDeviceSynchronize();
	set_initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);
	
	/* Print di debug riguardante lo stato dei nodi e i pivot iniziali
	
	HANDLE_ERROR(hipMemcpy(status_tmp, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemcpy(pivots_tmp, d_pivots, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
	for(int i = 0; i < num_nodes; i++){
		printf("status[%d] = %s, pivots[%d] = %d\n", i, from_status_to_string(status_tmp[i]), i, pivots_tmp[i]);
	} */

	HANDLE_ERROR(hipMemcpy(d_bw_status, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToDevice));

	// Si ripete il ciclo fino a quando tutti i nodi vengono eliminati
	*stop = false;
	*bw_stop = false;
    while (!*stop){
		// Forward + Backward reach
		DEBUG_MSG("Reach:" , "", DEBUG_FW_BW);
       	
		*stop = false;
		*bw_stop = false;
		HANDLE_ERROR(hipMemcpy(d_bw_status, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToDevice));
		while(!(*stop && *bw_stop)) {
			
			#pragma omp parallel sections if(num_nodes>OMP_MIN_NODES) num_threads(MAX_THREADS_OMP)
			{
				#pragma omp section 
				{
					if(!*stop){
						*stop = true;
						f_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream[1]>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, d_stop, h_get_fw_visited, h_get_fw_expanded, h_set_fw_visited, h_set_fw_expanded);
					}
				}
				#pragma omp section 
				{
					if(!*bw_stop){
						*bw_stop = true;
						f_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream[2]>>>(num_nodes, d_nodes_transpose, d_adjacency_list_transpose, d_pivots, d_bw_status, d_bw_stop, h_get_bw_visited, h_get_bw_expanded, h_set_bw_visited, h_set_bw_expanded);
					}
				}
			}

			hipStreamSynchronize(stream[1]);
			hipStreamSynchronize(stream[2]);
		}

		/*  Print di debug riguardante lo stato dei nodi e i pivot
		
		HANDLE_ERROR(hipMemcpy(status_tmp, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(bw_status_tmp, d_bw_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost));
		for(int i = 0; i < num_nodes; i++){
			printf("fw_status[%d] = %s, bw_status[%d] = %s\n", i, from_status_to_string(status_tmp[i]), i, from_status_to_string(bw_status_tmp[i]));
		}
		for(int i = 0; i < num_nodes; i++){
			printf("pivots[%d] = %d\n", i, pivots_tmp[i]);
		} */
 		
		bitwise_or_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_status, d_bw_status);
		hipDeviceSynchronize();

		// Update dei pivot
		DEBUG_MSG("Update:" , "", DEBUG_FW_BW);
		update_v6(num_nodes, d_pivots, d_status, d_colors, d_write_id_for_pivots, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		// Trimming per eliminare ulteriori nodi che non hanno più out-degree e in-degree diversi da 0
		if(!*stop){
			DEBUG_MSG("Trimming:" , "", DEBUG_FW_BW);
			trimming_v6(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
			*stop = false;
		}

		/* Print riguardante i nodi elimitati
		HANDLE_ERROR(hipMemcpy(status_tmp, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost));
		for(int i = 0; i < num_nodes; i++){
			printf("status[%d] = %s\n", i, from_status_to_string(status_tmp[i]));
		} */

		/* Print riguardante i nuovi pivot
		HANDLE_ERROR(hipMemcpy(pivots_tmp, d_pivots, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost));
		for(int i = 0; i < num_nodes; i++){
			printf("pivots[%d] = %d\n", i, pivots_tmp[i]);
		}
		printf("---------------------\n");

		hipDeviceSynchronize();
		HANDLE_ERROR(hipMemcpy(status_tmp, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		*/
    }
	
	#pragma omp parallel sections if(num_nodes>OMP_MIN_NODES) num_threads(MAX_THREADS_OMP)
	{
		#pragma omp section
		{
			hipHostFree(stop);
			hipHostFree(d_stop);
		}

		#pragma omp section
		{
			HANDLE_ERROR(hipFreeAsync(d_write_id_for_pivots, stream[0]));
		}

		#pragma omp section
		{
			HANDLE_ERROR(hipFreeAsync(d_colors, stream[1]));
		}

		#pragma omp section
		{
			hipHostFree(h_get_fw_visited);
			hipHostFree(h_get_bw_visited);
		}

		#pragma omp section
		{
			hipHostFree(h_set_fw_visited);
			hipHostFree(h_set_bw_visited);
		}

		#pragma omp section
		{
			hipHostFree(h_get_fw_expanded);
			hipHostFree(h_get_bw_expanded);
		}

		#pragma omp section
		{
			hipHostFree(h_set_fw_expanded);
			hipHostFree(h_set_bw_expanded);
			hipHostFree(h_set_bw_expanded);
		}
	}

	// Tramite fw_bw_ abbiamo ottenuto, per ogni nodo, il pivot della SCC a cui appartiene.
	// Allochiamo is_scc, che alla fine avrà per ogni nodo il pivot della sua SCC se la sua SCC è accettabile, altrimenti -1
	trim_u_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status);
	
	bool * d_is_scc;
    #pragma omp parallel sections if(num_nodes>OMP_MIN_NODES) num_threads(MAX_THREADS_OMP)
	{	
		#pragma omp section
		{
			HANDLE_ERROR(hipHostUnregister(adjacency_list_transpose));
			HANDLE_ERROR(hipFreeAsync(d_adjacency_list_transpose, stream[1]));
		}

		#pragma omp section
		{
			HANDLE_ERROR(hipHostUnregister(adjacency_list));
			HANDLE_ERROR(hipFreeAsync(d_adjacency_list, stream[2]));
		}

		#pragma omp section
		{
			HANDLE_ERROR(hipHostUnregister(nodes_transpose));
			HANDLE_ERROR(hipFreeAsync(d_nodes_transpose, stream[3]));
		}

		#pragma omp section
		{
			HANDLE_ERROR(hipHostUnregister(nodes));
			HANDLE_ERROR(hipFreeAsync(d_nodes, stream[4]));
		}

		#pragma omp section
		{	
			HANDLE_ERROR(hipMalloc((void**)&d_is_scc, num_nodes * sizeof(unsigned int)));
			trim_u_propagation<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status, d_is_scc);
		}
    }

	if(profiling){
		eliminate_trivial_scc<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, THREADS_PER_BLOCK*sizeof(unsigned int) + THREADS_PER_BLOCK*sizeof(bool)>>>(THREADS_PER_BLOCK, num_nodes, d_pivots, d_is_scc);
		
		bool result = is_there_an_scc(NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, num_nodes, d_is_scc);
		printf("%d\n", result);
	}else{
		// Nella versione naive, una funzione calcolava il numero di nodi di una SCC e poi "cancellava" quelli con un numero < 2.
		// La funzione è stata eliminata e is_scc_adjust si occupa di "cancellare" tali nodi senza doverli contare.
		// N.B. Per "cancellare" si intende assegnare ad un generico nodo v is_scc[v] = -1
		is_scc_adjust<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);
		hipDeviceSynchronize();
		is_scc_adjust_prop<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);

		unsigned int * pivots;
		char * final_status;

		pivots = (unsigned int*) malloc(num_nodes * sizeof(unsigned int));
		final_status = (char*) malloc(num_nodes * sizeof(char));

		hipMemcpy(pivots, d_pivots, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(final_status, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost);

		DEBUG_MSG("Number of SCCs found: ", count_distinct_scc(num_nodes, pivots, final_status), DEBUG_FINAL);

		free(final_status);
		free(pivots);
	}

	#pragma omp parallel if(num_nodes>OMP_MIN_NODES) num_threads(MAX_THREADS_OMP)
	{
		#pragma omp sections nowait
		{
			#pragma omp section 
			{
				HANDLE_ERROR(hipFree(d_is_scc));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipFree(d_status));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipFree(d_pivots));
			}

			#pragma omp section 
			{
				HANDLE_ERROR(hipHostUnregister(status));
			}
		}

		#pragma omp barrier

		#pragma omp for schedule(static)
		for (short i = 0; i < CUDA_STREAMS; i++) {
			hipStreamDestroy(stream[i]);
		}
	}


}