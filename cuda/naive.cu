#include "hip/hip_runtime.h"
#include "../utils/is_checked.cu"
#include "../utils/file2graph.cpp"
#include <cstring>
#include <hip/hip_runtime.h>
using namespace std;

#define DEBUG_F_KERNEL false
#define DEBUG_REACH false
#define DEBUG_TRIMMING_KERNEL false
#define DEBUG_TRIMMING false
#define DEBUG_UPDATE false
#define DEBUG_FW_BW false
#define DEBUG_MAIN false
#define DEBUG_FINAL true

static void handle_error(hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (handle_error( err, __FILE__, __LINE__ ))

__global__ void f_kernel(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, char * d_status, bool * d_stop, bool (*get_visited)(char *), bool (*get_expanded)(char *), void (*set_visited)(char *), void (*set_expanded)(char *)){
	// Esecuzione di un thread della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

    // Per ogni nodo
	if(v < num_nodes) {
        // Si controlla se non è stato eliminato E è stato eliminato E non è stato espanso
		if(!get_is_d_eliminated(&d_status[v]) && get_visited(&d_status[v]) && !get_expanded(&d_status[v])) {
            // Si segna come espanso
			set_expanded(&d_status[v]);

            // Per ogni nodo a cui punta
			for(int u = d_nodes[v]; u < d_nodes[v + 1]; u++) {	
				int dst = d_adjacency_list[u];

                // Si controlla se non è stato eliminato E se non è stato visitato E se il colore del nodo che punta corrisponde a quello del nodo puntato
				if(!get_is_d_eliminated(&d_status[dst]) && !get_visited(&d_status[dst]) && d_pivots[v] == d_pivots[dst]) {
                    // Setta il nodo puntato a visitato
					set_visited(&d_status[dst]);
                    // Permette di continuare il ciclo in reach, perchè si è trovato un altro nodo da visitare
					*d_stop = false;
				}
			}
		}
	}
}

void reach(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, char * d_status, bool (*get_visited)(char *), bool (*get_expanded)(char *), void (*set_visited)(char *), void (*set_expanded)(char *), const int t_per_blocks,  const int n_blocks) {
	// Esecuzione ricorsiva della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming

	bool stop, *d_stop;
	stop = false;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));
	
    // Si effettua la chiusura in avanti/indietro
    while(!stop) {
		HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
        f_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, d_stop, get_visited, get_expanded, set_visited, set_expanded);	
		HANDLE_ERROR(hipMemcpy(&stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
    }
	
	HANDLE_ERROR(hipFree(d_stop));
}

__global__ void trimming_kernel(int num_nodes, int * d_nodes, int * d_nodes_transpose, int * d_adjacency_list,  int * d_adjacency_list_transpose, char * d_status, bool * d_stop){
	// Esegue un'eliminazione di nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes) {
		if(!get_is_d_eliminated(&d_status[v])){
			// Se questo valore non verrà cambiato, allora il nodo verrà cancellato
			bool elim = true;

			bool forward = false;
			bool backward = false;
			
			// Nel caso un nodo abbia entrambi in_degree o out_degree diversi da 0, tra i soli nodi non eliminati, allora non va eliminato
			for(int u = d_nodes[v]; u < d_nodes[v+1]; u++){
				if(!get_is_d_eliminated(&d_status[d_adjacency_list[u]])) {
					forward = true;
				}
			}
			if(forward) {
				for(int u = d_nodes_transpose[v]; u < d_nodes_transpose[v+1]; u++){
					if(!get_is_d_eliminated(&d_status[d_adjacency_list_transpose[u]])) {
						backward = true;
					}
				}
			}
			if(backward) {
				elim = false;
			}

			if(elim){
				set_is_d_eliminated(&d_status[v]);
				*d_stop = false;
			}
		}
	}
}

void trimming(int num_nodes, int * d_nodes, int * d_nodes_transpose, int * d_adjacency_list, int * d_adjacency_list_transpose, char * d_status, const int n_blocks, const int t_per_blocks) {
	// Elimina iterativamente i nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming

	bool stop, *d_stop;
	stop = false;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));

    while(!stop) {
		HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
        trimming_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, d_stop);
		HANDLE_ERROR(hipMemcpy(&stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
    }

	HANDLE_ERROR(hipFree(d_stop));
}

__global__ void set_colors(int num_nodes, char * d_status, int * d_pivots, int * d_colors, long * d_write_id_for_pivots, bool * d_stop){
	// Esegue l'update dei valori del pivot facendo una race, scrivendo il "colore" di una serie di pivot in array simultaneamente
	// @param:	pivots						= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated				= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited				= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited				= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: d_write_id_for_pivots		= Lista che conterrà, nelle posizione identificate dai colori appena calcolati, i nuovi pivot da assegnare
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes) {
		if(get_is_d_eliminated(&d_status[v])){
			d_pivots[v] = v;
		}

		const bool fw_visitated = get_is_d_fw_visited(&d_status[v]);
		const bool bw_visitated = get_is_d_bw_visited(&d_status[v]);
		
		if(fw_visitated == bw_visitated && fw_visitated == true){
			d_colors[v] = 4 * d_pivots[v];
		} else {
			if(fw_visitated != bw_visitated && fw_visitated == true){
				d_colors[v] = 4 * d_pivots[v] + 1;
			}else if(fw_visitated != bw_visitated && fw_visitated == false){
				d_colors[v] = 4 * d_pivots[v] + 2;
			}else if(fw_visitated == bw_visitated && fw_visitated == false){
				d_colors[v] = 4 * d_pivots[v] + 3;				
			}
				
			if(!get_is_d_eliminated(&d_status[v])){
				*d_stop = false;
			}
		}
		d_write_id_for_pivots[d_colors[v]] = v;
	}
}

__global__ void set_race_winners(int num_nodes, char * d_status, int * d_pivots, int * d_colors, long * d_write_id_for_pivots){
	// Ottenuti i valori della race, si vanno ad impostare i nuovi pivot
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update

	
	int v = threadIdx.x + blockIdx.x * blockDim.x;
	if(v < num_nodes) {
		// Se il nodo è stato eliminato, allora il suo pivot è per forza se stesso
		if(get_is_d_eliminated(&d_status[v])){
			d_pivots[v] = v;
		}else{
			d_pivots[v] = d_write_id_for_pivots[d_colors[v]];
			set_is_d_fw_visited(&d_status[d_pivots[v]]);
			set_is_d_bw_visited(&d_status[d_pivots[v]]);
		}
	}
}

__global__ void initialize_pivot(int num_nodes, int * d_pivots, char * d_status) {
	// Scelta iniziale del primo pivot, basandosi sui nodi cancellati inizialmente
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, avente come pivot un nodo non cancellato
	//          fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no. A questo punto l'unico nodo visitato è il solo pivot scelto
	//          bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no. A questo punto l'unico nodo visitato è il solo pivot scelto

	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes){
		__shared__ int chosen_pivot;
		if(!get_is_d_eliminated(&d_status[v])){
			chosen_pivot = v;
		}

		// Sincronizziamo qui i thread per inizializzare questi array: lanciare un altro thread
		// solo per inizializzare gli array potrebbe risultare più pesante che farlo qui
		__syncthreads();

		d_pivots[v] = chosen_pivot;
		set_is_d_fw_visited(&d_status[d_pivots[v]]);
		set_is_d_bw_visited(&d_status[d_pivots[v]]);
	}
}

void update(int num_nodes, int * d_pivots, char * d_status, long * d_write_id_for_pivots, int * d_colors, bool * stop, const int n_blocks, const int t_per_blocks) {
	// Esegue l'update dei valori del pivot facendo una race
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update


	bool *d_stop;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));
	
	HANDLE_ERROR(hipMemset(d_write_id_for_pivots, -1, 4 * num_nodes * sizeof(long)));
	HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
	
	// Dai paper:
	// These subgraphs are 
	// 		1) the strongly connected component with the pivot;
	// 		2) the subgraph given by vertices in the forward closure but not in the backward closure; 
	// 		3) the subgraph given by vertices in the backward closure but not in the forward closure;
	// 		4) the subgraph given by vertices that are neither in the forward nor in the backward closure.
	
	// The subgraphs that do not contain the pivot form three independent instances of the same problem, and therefore, 
	// they are recursively processed in parallel with the same algorithm
	
	set_colors<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots, d_stop);
	
	HANDLE_ERROR(hipMemcpy(stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(d_stop));

	// Setto i valori dei pivot che hanno vinto la race
	// Se sono stati eliminati, allora setta il valore dello stesso nodo 
	set_race_winners<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots);
}

__global__ void trim_u_kernel(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, char * d_status, int * d_is_scc){
	// Setta i pivot delle SCC uguale a -1 se questi ricevono archi da nodi u
	// param: 	pivots = 	Lista che per ogni 'v' dice il valore del pivot della SCC
	// 			is_scc =	Lista copia di pivots
	// @return:	is_scc =	Lista contenente i pivot delle SCC, però i pivot delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes){
		if(get_is_d_u(&d_status[v])){
			for(int u = d_nodes[v]; u < d_nodes[v+1]; ++u) {
				if(d_pivots[v] != d_pivots[d_adjacency_list[u]]) {
					d_is_scc[d_pivots[d_adjacency_list[u]]] = -1;
				}
			}
		}
	}

}

__global__ void trim_u_propagation(int num_nodes, int * d_pivots, int * d_is_scc) {
	// Se alcuni pivot sono settati a -1, per la cancellazione dovuta a collegamenti con nodi u, 
	// propaga la cancellazione agli altri membri della SCC
	// param: 	pivots = 	Lista contenente i pivot delle SCC
	// 			is_scc =	Lista contenente i pivot delle SCC, però i pivot delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1
	// @return:	is_scc =	Lista contenente i pivot delle SCC, però i pivot e gli altri nodi delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1

	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if (v < num_nodes)
		d_is_scc[v] = d_is_scc[d_pivots[v]];
}

__global__ void calculate_more_than_one(int num_nodes, int * d_more_than_one_dev, int * is_scc_dev) {
	// Trova il numero di elementi nella SCC
	// @param: is_scc =	Lista contenente i pivot delle SCC, però i pivot e gli altri nodi delle SCC 
	// 					che ricevono archi da nodi u sono settati a -1
	// @return:	more_than_one = 	Lista che per ogni nodo 'v' dice se questo è un pivot.
	// 								Se 'v' è pivot: 	more_than_one[v] = numero di elementi nella sua SCC,
	// 								Se 'v' non è pivot:	more_than_one[v] = 1

	int u = threadIdx.x + blockIdx.x * blockDim.x;

	if (u < num_nodes){
		if(is_scc_dev[u] != -1){
			// atomicAdd può essere migliorato -> Simile al problema dell'istogramma
			atomicAdd(&d_more_than_one_dev[is_scc_dev[u]], 1);
		}
	}
}

__global__ void is_scc_adjust(int num_nodes, int * more_than_one_dev, int * is_scc_dev) {
	// Restituisce una lista che dice se il nodo 'v' fa parte di una SCC
	// @param: more_than_one = 	Lista che per ogni nodo 'v' dice se questo è un pivot.
	// 							Se 'v' è pivot: 								more_than_one[v] = numero di elementi nella sua SCC,
	// 							Se 'v' non è pivot, ma fa parte di una SCC:		more_than_one[v] = 0
	// 							Se 'v' non è pivot e non fa parte di una SCC:	more_than_one[v] = 0
	// @return: is_scc =	Lista che per ogni nodo 'v' dice se questo fa parte di una SCC.
	// 						Se fa parte di una SCC: 	is_scc[v] = valore del pivot,
	// 						Se non fa parte di una SCC:	is_scc[v] = -1

	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if (v < num_nodes){
		if(more_than_one_dev[v] == 1)
			is_scc_dev[v] = -1;
	}
}

int count_distinct_scc(int is_scc[], int num_nodes){
	// Restituisce il numero di SCC valide presenti nell'array is_scc
	// Questa funzione non viene parallelizzata poiché utilizzata solamente per verificare la correttezza del risultato
	// @param:  is_scc 	= 	Lista contenente le SCC valide trovate
	// @return: res    	=	Numero di SCC valide diverse

    int res = 0;
 
    // Per tutti gli elementi dell'array
    for (int i = 1; i < num_nodes; i++) {
        int j = 0;
        for (j = 0; j < i; j++)
            if (is_scc[i] == is_scc[j])
                break;
 
        // Se non è già stato contato, contalo
        if (i == j)
            res++;
    }
    return res;
}

int main(int argc, char ** argv) {
	// Impostazione del device
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

    if (argc != 2) {
		cout << " Invalid Usage !! Usage is ./main.out <graph_input_file> \n";
		return -1;
	}

	int num_nodes, num_edges;
    int * nodes, * adjacency_list, * nodes_transpose, * adjacency_list_transpose, * is_scc;
	char * status;

    create_graph_from_filename(argv[1], num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status);

	if(DEBUG_MAIN){
		for (int i = 0; i < num_nodes; i++)
			DEBUG_MSG("nodes[" + to_string(i) + "] = ", nodes[i], DEBUG_MAIN);
		for (int i = 0; i < num_edges; i++)
			DEBUG_MSG("adjacency_list[" + to_string(i) + "] = ", adjacency_list[i], DEBUG_MAIN);
		for (int i = 0; i < num_nodes; i++)
			DEBUG_MSG("nodes_transpose[" + to_string(i) + "] = ", nodes_transpose[i], DEBUG_MAIN);
		for (int i = 0; i < num_edges; i++)
			DEBUG_MSG("adjacency_list_transpose[" + to_string(i) + "] = ", adjacency_list_transpose[i], DEBUG_MAIN);
	}

	const int THREADS_PER_BLOCK = prop.maxThreadsPerBlock;
	const int NUMBER_OF_BLOCKS = num_nodes / THREADS_PER_BLOCK + (num_nodes % THREADS_PER_BLOCK == 0 ? 0 : 1);

	// Dichiarazioni di variabili device
	int * d_is_scc, * d_more_than_one, * d_nodes, * d_adjacency_list, * d_nodes_transpose, * d_adjacency_list_transpose, * d_pivots, * d_colors;
	char * d_status;
	long * d_write_id_for_pivots;

	HANDLE_ERROR(hipMalloc((void**)&d_nodes, (num_nodes+1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_nodes_transpose, (num_nodes+1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_adjacency_list, num_edges * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_adjacency_list_transpose, num_edges * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_pivots, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_colors, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_status, num_nodes * sizeof(char)));

	HANDLE_ERROR(hipMalloc((void**)&d_write_id_for_pivots, 4 * num_nodes * sizeof(long)));

	// Le strutture principali le copiamo nel device già qui, visto che non verranno mai modificate
	HANDLE_ERROR(hipMemcpy(d_nodes, nodes, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_adjacency_list, adjacency_list, num_edges * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_nodes_transpose, nodes_transpose, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_adjacency_list_transpose, adjacency_list_transpose, num_edges * sizeof(int), hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(d_status, status, num_nodes * sizeof(char), hipMemcpyHostToDevice));

	// Inizializzazione e copia delle funzioni device che verranno passate tramite parametro.
	// Utilizzando le funzioni in questo modo, anche se apparentemente verboso, permette di ottenere meno codice duplicato:
	// infatti, se non fosse per queste variabili, si sarebbe dovuto duplicare l'f_kernel e il reach per averne uno per la forward e uno per la backward.
	get_status h_get_fw_visited, h_get_bw_visited, h_get_fw_expanded, h_get_bw_expanded;
	set_status h_set_fw_visited, h_set_bw_visited, h_set_fw_expanded, h_set_bw_expanded;

	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_fw_visited, HIP_SYMBOL(dev_get_fw_visited), sizeof(get_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_bw_visited, HIP_SYMBOL(dev_get_bw_visited), sizeof(get_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_fw_expanded, HIP_SYMBOL(dev_get_fw_expanded), sizeof(get_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_get_bw_expanded, HIP_SYMBOL(dev_get_bw_expanded), sizeof(get_status)));

	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_fw_visited, HIP_SYMBOL(dev_set_fw_visited), sizeof(set_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_bw_visited, HIP_SYMBOL(dev_set_bw_visited), sizeof(set_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_fw_expanded, HIP_SYMBOL(dev_set_fw_expanded), sizeof(set_status)));
	HANDLE_ERROR(hipMemcpyFromSymbol(&h_set_bw_expanded, HIP_SYMBOL(dev_set_fw_expanded), sizeof(set_status)));
	
	// Primo trimming per eliminare i nodi che, dopo la cancellazione dei nodi non in U,
	// non avevano più out-degree e in-degree diverso da 0
	trimming(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, THREADS_PER_BLOCK, NUMBER_OF_BLOCKS);
	
	// Si fanno competere i thread per scelgliere un nodo che farà da pivot, a patto che quest'ultimo sia non eliminato
	initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);
	
    bool stop = false;
	
	// Si ripete il ciclo fino a quando tutti i nodi vengono eliminati
    while (!stop){
		// Forward reach
		DEBUG_MSG("Forward reach:" , "", DEBUG_FW_BW);
        reach(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, h_get_fw_visited, h_get_fw_expanded, h_set_fw_visited, h_set_fw_expanded, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
		
		// Backward reach
        DEBUG_MSG("Backward reach:" , "", DEBUG_FW_BW);
		reach(num_nodes, d_nodes_transpose, d_adjacency_list_transpose, d_pivots, d_status, h_get_bw_visited, h_get_bw_expanded, h_set_bw_visited, h_set_bw_expanded, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		// Trimming per eliminare ulteriori nodi che non hanno più out-degree e in-degree diversi da 0
		DEBUG_MSG("Trimming:" , "", DEBUG_FW_BW);
        trimming(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, THREADS_PER_BLOCK, NUMBER_OF_BLOCKS);

		// Update dei pivot
		DEBUG_MSG("Update:" , "", DEBUG_FW_BW);
		update(num_nodes, d_pivots, d_status, d_write_id_for_pivots, d_colors, &stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
    }
	
	//Disallocamento della memoria iniziale
	HANDLE_ERROR(hipFree(d_write_id_for_pivots));
	
	// Tramite fw_bw_ abbiamo ottenuto, per ogni nodo, il pivot della SCC a cui appartiene.
	// Allochiamo is_scc, che alla fine avrà per ogni nodo il pivot della sua SCC se la sua SCC è accettabile, altrimenti -1
	
	// Per iniziare le assegnamo gli stessi valori di pivots, che verranno modificati in seguito
	is_scc = (int*) malloc(num_nodes * sizeof(int));
	
	// Allochiamo more_than_one, che per ogni nodo che fa da pivot viene assegnato un contatore, il quale conta quante volte appare tale pivot
	// Se appare solo una volta, allora il nodo non fa parte di nessuna SCC
	HANDLE_ERROR(hipMalloc((void**)&d_is_scc, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_more_than_one, num_nodes * sizeof(int)));
	
	HANDLE_ERROR(hipMemcpy(d_is_scc, d_pivots, num_nodes * sizeof(int), hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemset(d_more_than_one, 0, num_nodes * sizeof(int)));

	trim_u_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, d_is_scc);
	
	HANDLE_ERROR(hipFree(d_adjacency_list_transpose));
	HANDLE_ERROR(hipFree(d_adjacency_list));
	HANDLE_ERROR(hipFree(d_nodes_transpose));
	HANDLE_ERROR(hipFree(d_nodes));
	
	trim_u_propagation<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_is_scc);

	HANDLE_ERROR(hipFree(d_pivots));

	calculate_more_than_one<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_more_than_one, d_is_scc);
	is_scc_adjust<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_more_than_one, d_is_scc);
	
	HANDLE_ERROR(hipFree(d_more_than_one));
	HANDLE_ERROR(hipFree(d_status));

	HANDLE_ERROR(hipMemcpy(is_scc, d_is_scc, num_nodes * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(d_is_scc));

	for (int i = 0; i < num_nodes; i++)
        DEBUG_MSG("is_scc[" + to_string(i) + "] = ", is_scc[i], false);

	DEBUG_MSG("Number of SCCs found: ", count_distinct_scc(is_scc, num_nodes), DEBUG_FINAL);
}