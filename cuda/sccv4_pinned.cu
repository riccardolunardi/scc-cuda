#include "../utils/is_checked.cu"
#include "../utils/file2graph.cpp"
#include "scc_operations.cu"
#include <cstring>
#include <hip/hip_runtime.h>
#include <set>
using namespace std;

#define DEBUG_F_KERNEL false
#define DEBUG_REACH false
#define DEBUG_TRIMMING_KERNEL false
#define DEBUG_TRIMMING false
#define DEBUG_UPDATE false
#define DEBUG_FW_BW false
#define DEBUG_MAIN false
#define DEBUG_FINAL true

#define CUDA_STREAMS 9

/*

VERSIONE DEL CODICE CUDA: SCCv4 - Pinned Memory

Questa versione del codice è un miglioramento della versione naive, in quanto si è andato a ottimizzare molti aspetti del codice:
- Le operazioni sulla memoria adesso vengono eseguite su stream diversi, sincronizzando il codice quando necessario
- Creazione di un'unica variabile "stop" da usare nei vari passaggi principali: si evita ogni volta una nuova allocazione
- Utilizzo dei registri all'interno dei kernel, per velocizzare le operazioni
- Utilizzo di un doppio shift, rimipazzando la moltiplicazione per 4
- set_colors e set_race_winners sono stati uniti, evitando il lancio di un kernel non essenziale
- Rimozione dell'array colors: tramite la programmazione parallela possiamo usare una sola variabile
- Alcune operazioni binarie su "status" sono state unite in una sola (es. 100 | 010 | 001 === 110 | 001 )
- Utilizzo di unsigned int. Nonostante non vengano migliorate direttamente le performace, c'è la possibilità di poter elaborare un numero più alto di nodi/archi
  senza dove aumentare lo spazio utilzzato in memoria

N.B.
- Non è possibile fare uso della memoria shared visto il tipo operazioni eseguite: spesso, ad esempio tramite i pivots, il codice "salta"
  da una posizione all'altra per accedere ai vari nodi. Visto che il massimo possibile sarebbe solo di salvare un frammento delle liste in memoria shared ed
  è possibile anticipare dove si andrà a leggere gli array, è impossibile farne uso.
- Non è possibile l'esecuzione in contemporanea (su stream diversi) di kernel diversi. Per funzionare correttamente ogni kernel deve ricevere i risultati di quello prima.
  L'unico caso che sarebbe possibile è quello del forward e backward reach, se non fosse che entrambi modificano l'array "status" e ci sarebbe una race condition non favorevole

*/

void reach_v4(unsigned int const num_nodes, unsigned int * d_nodes, unsigned int * d_adjacency_list, unsigned int * d_pivots, char * d_status, bool (*get_visited)(char *), bool (*get_expanded)(char *), void (*set_visited)(char *), void (*set_expanded)(char *), bool * stop, bool * d_stop, const unsigned int n_blocks, const unsigned int t_per_blocks) {
	// Esecuzione ricorsiva della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming

	*stop = false;

    // Si effettua la chiusura in avanti/indietro
    while(!*stop) {
		*stop = true;
        f_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, d_stop, get_visited, get_expanded, set_visited, set_expanded);
		hipDeviceSynchronize();
    }
}

void trimming_v4(unsigned int const num_nodes, unsigned int * d_nodes, unsigned int * d_nodes_transpose, unsigned int * d_adjacency_list, unsigned int * d_adjacency_list_transpose, char * d_status, bool * stop, bool * d_stop, const unsigned int n_blocks, const unsigned int t_per_blocks) {
	// Elimina iterativamente i nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming

	*stop = false;
    while(!*stop) {
		*stop = true;
        trimming_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, d_stop);
		// Dobbiamo aspettare che la copia venga effettuata anche se è mappata
		hipDeviceSynchronize();
    }
}

void update_v4(unsigned int const num_nodes, unsigned int * d_pivots, char * d_status,  unsigned int * d_colors, unsigned long * d_write_id_for_pivots, bool * stop, bool * d_stop, const unsigned int n_blocks, const unsigned int t_per_blocks) {
	// Esegue l'update dei valori del pivot facendo una race
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update
	
	*d_stop = true;
	// Dai paper:
	// These subgraphs are 
	// 		1) the strongly connected component with the pivot;
	// 		2) the subgraph given by vertices in the forward closure but not in the backward closure; 
	// 		3) the subgraph given by vertices in the backward closure but not in the forward closure;
	// 		4) the subgraph given by vertices that are neither in the forward nor in the backward closure.
	
	// The subgraphs that do not contain the pivot form three independent instances of the same problem, and therefore, 
	// they are recursively processed in parallel with the same algorithm
	
	// Setto i valori dei pivot che hanno vinto la race
	// Se sono stati eliminati, allora setta il valore dello stesso nodo 
	set_colors<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots, d_stop);
	hipDeviceSynchronize();
	set_new_pivots<<<n_blocks, t_per_blocks>>>(num_nodes, d_status, d_pivots, d_colors, d_write_id_for_pivots);
	hipDeviceSynchronize();
	/* HANDLE_ERROR(hipMemcpy(main_stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipMemset(d_stop, false, sizeof(bool))); */
}

void routine_v4(const bool profiling, unsigned int num_nodes, unsigned int num_edges, unsigned * nodes, unsigned * adjacency_list, unsigned * nodes_transpose, unsigned * adjacency_list_transpose, char * status) {
	// Impostazione del device
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	hipSetDeviceFlags(hipDeviceMapHost);

	bool * d_stop, * stop;

	// Dichiarazioni di variabili device
	unsigned int * d_nodes, * d_adjacency_list, * d_nodes_transpose, * d_adjacency_list_transpose, * d_pivots, * d_colors;
	char * d_status;
	unsigned long * d_write_id_for_pivots;

	HANDLE_ERROR(hipHostRegister(nodes, (num_nodes+1) * sizeof(unsigned int), hipHostRegisterDefault));
	HANDLE_ERROR(hipHostRegister(adjacency_list, num_edges * sizeof(unsigned int), hipHostRegisterDefault));
	HANDLE_ERROR(hipHostRegister(nodes_transpose, (num_nodes+1) * sizeof(unsigned int), hipHostRegisterDefault));
	HANDLE_ERROR(hipHostRegister(adjacency_list_transpose, num_edges * sizeof(unsigned int), hipHostRegisterDefault));
	HANDLE_ERROR(hipHostRegister(status, (num_nodes+1) * sizeof(char), hipHostRegisterDefault));

	/* if(DEBUG_MAIN){
		for (unsigned int i = 0; i < num_nodes; i++)
			DEBUG_MSG("nodes[" + to_string(i) + "] = ", nodes[i], DEBUG_MAIN);
		for (unsigned int i = 0; i < num_edges; i++)
			DEBUG_MSG("adjacency_list[" + to_string(i) + "] = ", adjacency_list[i], DEBUG_MAIN);
		for (unsigned int i = 0; i < num_nodes; i++)
			DEBUG_MSG("nodes_transpose[" + to_string(i) + "] = ", nodes_transpose[i], DEBUG_MAIN);
		for (unsigned int i = 0; i < num_edges; i++)
			DEBUG_MSG("adjacency_list_transpose[" + to_string(i) + "] = ", adjacency_list_transpose[i], DEBUG_MAIN);
	} */

	const unsigned int THREADS_PER_BLOCK = prop.maxThreadsPerBlock;
	const unsigned int NUMBER_OF_BLOCKS = num_nodes / THREADS_PER_BLOCK + (num_nodes % THREADS_PER_BLOCK == 0 ? 0 : 1);

	// Inizializzazione e copia delle funzioni device che verranno passate tramite parametro.
	// Utilizzando le funzioni in questo modo, anche se apparentemente verboso, permette di ottenere meno codice duplicato:
	// infatti, se non fosse per queste variabili, si sarebbe dovuto duplicare l'f_kernel e il reach per averne uno per la forward e uno per la backward.
	get_status h_get_fw_visited, h_get_bw_visited, h_get_fw_expanded, h_get_bw_expanded;
	set_status h_set_fw_visited, h_set_bw_visited, h_set_fw_expanded, h_set_bw_expanded;

	hipStream_t stream[CUDA_STREAMS];
	for (unsigned int i=0; i<CUDA_STREAMS; ++i){
		hipStreamCreate(&stream[i]);
	}

	HANDLE_ERROR(hipMallocAsync((void**)&d_write_id_for_pivots, 4 * num_nodes * sizeof(unsigned long), stream[0]));
	HANDLE_ERROR(hipMallocAsync((void**)&d_colors, num_nodes * sizeof(unsigned int), stream[0]));

	HANDLE_ERROR(hipMallocAsync((void**)&d_adjacency_list, num_edges * sizeof(unsigned int), stream[1]));
	HANDLE_ERROR(hipMallocAsync((void**)&d_adjacency_list_transpose, num_edges * sizeof(unsigned int), stream[2]));
	HANDLE_ERROR(hipMallocAsync((void**)&d_nodes, (num_nodes+1) * sizeof(unsigned int), stream[3]));
	HANDLE_ERROR(hipMallocAsync((void**)&d_nodes_transpose, (num_nodes+1) * sizeof(unsigned int), stream[4]));
	HANDLE_ERROR(hipMallocAsync((void**)&d_pivots, num_nodes * sizeof(unsigned int), stream[5]));
	HANDLE_ERROR(hipMallocAsync((void**)&d_status, (num_nodes+1) * sizeof(char), stream[6]));

	// Le strutture principali le copiamo nel device già qui, visto che non verranno mai modificate
	HANDLE_ERROR(hipMemcpyAsync(d_adjacency_list, adjacency_list, num_edges * sizeof(unsigned int), hipMemcpyHostToDevice, stream[1]));
	HANDLE_ERROR(hipMemcpyAsync(d_adjacency_list_transpose, adjacency_list_transpose, num_edges * sizeof(unsigned int), hipMemcpyHostToDevice, stream[2]));
	HANDLE_ERROR(hipMemcpyAsync(d_nodes, nodes, (num_nodes+1) * sizeof(unsigned int), hipMemcpyHostToDevice, stream[3]));
	HANDLE_ERROR(hipMemcpyAsync(d_nodes_transpose, nodes_transpose, (num_nodes+1) * sizeof(unsigned int), hipMemcpyHostToDevice, stream[4]));
	HANDLE_ERROR(hipMemcpyAsync(d_status, status, (num_nodes+1) * sizeof(char), hipMemcpyHostToDevice, stream[6]));

	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_fw_visited, HIP_SYMBOL(dev_get_fw_visited), sizeof(get_status), 0, hipMemcpyDefault, stream[0]));
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_bw_visited, HIP_SYMBOL(dev_get_bw_visited), sizeof(get_status), 0, hipMemcpyDefault, stream[5]));
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_fw_expanded, HIP_SYMBOL(dev_get_fw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[1]));
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_get_bw_expanded, HIP_SYMBOL(dev_get_bw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[2]));
	
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_fw_visited, HIP_SYMBOL(dev_set_fw_visited), sizeof(set_status), 0, hipMemcpyDefault, stream[3]));
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_bw_visited, HIP_SYMBOL(dev_set_bw_visited), sizeof(set_status), 0, hipMemcpyDefault, stream[4]));
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_fw_expanded, HIP_SYMBOL(dev_set_fw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[5]));
	HANDLE_ERROR(hipMemcpyFromSymbolAsync(&h_set_bw_expanded, HIP_SYMBOL(dev_set_bw_expanded), sizeof(get_status), 0, hipMemcpyDefault, stream[6]));

	HANDLE_ERROR(hipHostAlloc(&stop, sizeof(bool), hipHostMallocMapped));
	HANDLE_ERROR(hipHostGetDevicePointer(&d_stop, stop, 0));

	hipStreamSynchronize(stream[1]);
	hipStreamSynchronize(stream[2]);
	hipStreamSynchronize(stream[3]);
	hipStreamSynchronize(stream[4]);
	hipStreamSynchronize(stream[6]);
	
	// Primo trimming per eliminare i nodi che, dopo la cancellazione dei nodi non in U,
	// non avevano più out-degree e in-degree diverso da 0
	*stop = false;
    while(!*stop) {
		*stop = true;
        trimming_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream[1]>>>(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, d_stop);
    }
	
	// Sincronizzazione implicita perché si utilizza il default stream
	// Si fanno competere i thread per scelgliere un nodo che farà da pivot, a patto che quest'ultimo sia non eliminato
	initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);
	hipDeviceSynchronize();
	set_initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_status);

	// Si ripete il ciclo fino a quando tutti i nodi vengono eliminati
	*stop = false;
    while (!*stop){
		// Forward reach
		DEBUG_MSG("Forward reach:" , "", DEBUG_FW_BW);
        reach_v4(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status, h_get_fw_visited, h_get_fw_expanded, h_set_fw_visited, h_set_fw_expanded, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
		
		// Backward reach
        DEBUG_MSG("Backward reach:" , "", DEBUG_FW_BW);
		reach_v4(num_nodes, d_nodes_transpose, d_adjacency_list_transpose, d_pivots, d_status, h_get_bw_visited, h_get_bw_expanded, h_set_bw_visited, h_set_bw_expanded, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		// Trimming per eliminare ulteriori nodi che non hanno più out-degree e in-degree diversi da 0
		//DEBUG_MSG("Trimming:" , "", DEBUG_FW_BW);
        //trimming_v4(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
		
		// Update dei pivot
		DEBUG_MSG("Update:" , "", DEBUG_FW_BW);
		update_v4(num_nodes, d_pivots, d_status, d_colors, d_write_id_for_pivots, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		if(!*stop){
			DEBUG_MSG("Trimming:" , "", DEBUG_FW_BW);
			trimming_v4(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_status, stop, d_stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
			*stop = false;
		}
    }

	hipHostFree(stop);
	hipHostFree(d_stop);
	HANDLE_ERROR(hipFreeAsync(d_write_id_for_pivots, stream[0]));
	HANDLE_ERROR(hipFreeAsync(d_colors, stream[0]));
	
	// Tramite fw_bw_ abbiamo ottenuto, per ogni nodo, il pivot della SCC a cui appartiene.
	// Allochiamo is_scc, che alla fine avrà per ogni nodo il pivot della sua SCC se la sua SCC è accettabile, altrimenti -1
	trim_u_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream[6]>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_status);
	
	HANDLE_ERROR(hipFreeAsync(d_adjacency_list_transpose, stream[1]));
	HANDLE_ERROR(hipFreeAsync(d_adjacency_list, stream[2]));
	HANDLE_ERROR(hipFreeAsync(d_nodes_transpose, stream[3]));
	HANDLE_ERROR(hipFreeAsync(d_nodes, stream[4]));
	
	hipStreamSynchronize(stream[6]);

	bool * d_is_scc;
	HANDLE_ERROR(hipMalloc((void**)&d_is_scc, num_nodes * sizeof(unsigned int)));
	trim_u_propagation<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream[6]>>>(num_nodes, d_pivots, d_status, d_is_scc);

	hipStreamSynchronize(stream[6]);

	if(profiling){
		eliminate_trivial_scc<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, THREADS_PER_BLOCK*sizeof(unsigned int) + THREADS_PER_BLOCK*sizeof(bool)>>>(THREADS_PER_BLOCK, num_nodes, d_pivots, d_is_scc);
		hipDeviceSynchronize();
		
		bool result = is_there_an_scc(NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, num_nodes, d_is_scc);
		printf("%d\n", result);
	}else{
		// Nella versione naive, una funzione calcolava il numero di nodi di una SCC e poi "cancellava" quelli con un numero < 2.
		// La funzione è stata eliminata e is_scc_adjust si occupa di "cancellare" tali nodi senza doverli contare.
		// N.B. Per "cancellare" si intende assegnare ad un generico nodo v is_scc[v] = -1
		is_scc_adjust<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK, 0, stream[6]>>>(num_nodes, d_pivots, d_status);
		hipDeviceSynchronize();

		// Questa sezione di codice è temporanea, verrà rimossa al momento del test
		unsigned int * pivots;
		char * final_status;

		pivots = (unsigned int*) malloc(num_nodes * sizeof(unsigned int));
		final_status = (char*) malloc(num_nodes * sizeof(char));

		hipMemcpy(pivots, d_pivots, num_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(final_status, d_status, num_nodes * sizeof(char), hipMemcpyDeviceToHost);

		DEBUG_MSG("Number of SCCs found: ", count_distinct_scc(num_nodes, pivots, final_status), DEBUG_FINAL);

		HANDLE_ERROR(hipFree(d_pivots));
		HANDLE_ERROR(hipFree(d_status));
		free(final_status);
		free(pivots);
	}


	// Da scommentare una volta finito il progetto
	//HANDLE_ERROR(hipFreeAsync(d_pivots, stream[0]));
	//HANDLE_ERROR(hipFreeAsync(d_status, stream[1]));

	hipHostFree(h_get_fw_visited);
	hipHostFree(h_get_bw_visited);
	hipHostFree(h_set_fw_visited);
	hipHostFree(h_set_bw_visited);

	hipHostFree(h_get_fw_expanded);
	hipHostFree(h_get_bw_expanded);
	hipHostFree(h_set_fw_expanded);
	hipHostFree(h_set_bw_expanded);

	HANDLE_ERROR(hipHostUnregister(nodes));
	HANDLE_ERROR(hipHostUnregister(nodes_transpose));
	HANDLE_ERROR(hipHostUnregister(adjacency_list));
	HANDLE_ERROR(hipHostUnregister(adjacency_list_transpose));
	HANDLE_ERROR(hipHostUnregister(status));

	hipDeviceSynchronize();

	for (unsigned int i=0; i<CUDA_STREAMS; ++i){
		hipStreamDestroy(stream[i]);
	}
}