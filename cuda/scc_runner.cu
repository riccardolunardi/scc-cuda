#include "hip/hip_runtime.h"
#include "../main.cpp"
#include "sccv1_naive.cu"
#include "sccv2_status.cu"
#include "sccv3_streams.cu"
#include "sccv4_pinned.cu"
#include "sccv5_openmp.cu"
#include "sccv6_optreach.cu"
#include "sccv7_optreach.cu"
#include <chrono>
#include <vector>
using namespace std;

#ifndef WARMUP
#define WARMUP 5
#endif WARMUP

double calculateStandardDeviation(double mean, int n, double numbers[]) {
	// Calculate the sum of squared differences
	double sumSquaredDifferences = 0;
	for (int i = 0; i < n; i++) {
		double difference = numbers[i] - mean;
		sumSquaredDifferences += difference * difference;
	}

	// Calculate the standard deviation
	return sqrt(sumSquaredDifferences / n);
}

vector<double> common_routine(void (*routine_runner)(unsigned int, unsigned int, unsigned int*, unsigned int*, unsigned int*, unsigned int*, char * ), const unsigned int num_nodes, const unsigned int num_edges, unsigned int * nodes, unsigned int * adjacency_list, unsigned int * nodes_transpose, unsigned int * adjacency_list_transpose, char * status, char * og_status, const int repeat) {
	// Call the function passed as an argument
	vector<double> executionTimes;

	for(short i=0; i<repeat + WARMUP; i++){
		memcpy(status, og_status, num_nodes);

		auto start = chrono::high_resolution_clock::now();
		routine_runner(num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status);
		auto end = chrono::high_resolution_clock::now();

		if (WARMUP < i) {
			executionTimes.push_back(chrono::duration<double, milli>(end - start).count());
		}

		hipDeviceReset();
	}

	return executionTimes;
}

void print_benchmark(const vector<double> executionTimes) {
	// Calculate the mean
	if (WARMUP == 0) {
		cout << endl << "Warmup is disabled, execution times are not printed" << endl;
	}else{
		double sum = 0;
		for (double t : executionTimes) {
			sum += t;
		}
		double mean = sum / executionTimes.size();

		// Calculate the sum of squared differences
		double sumSquaredDifferences = 0;
		for (double t : executionTimes) {
			double difference = t - mean;
			sumSquaredDifferences += difference * difference;
		}

		// Calculate the standard deviation
		double standardDeviation = sqrt(sumSquaredDifferences / executionTimes.size());

		cout << endl << mean << "," << standardDeviation << endl;
	}

}

int main(unsigned int argc, char ** argv) {
    if (argc < 3) {
		cout << "Invalid Usage !! Usage is ./main.out <graph_input_file> <number_of_repetition>\n";
		return -1;
	}

	const int repeat = atoi(argv[2]);

	// Inizializzazione di struttrure dati per le versioni > 1
	unsigned num_nodes, num_edges;
    unsigned * nodes, * adjacency_list, * nodes_transpose, * adjacency_list_transpose;
	char * og_status;
	
	printf("Lettura del file %s...\n", argv[1]);
	create_graph_from_filename(argv[1], num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, og_status);

	printf("Number of nodes: %d\n", num_nodes);

	// Inizializzazione di struttrure dati per la versione 1
	int num_nodes_v1, num_edges_v1;
    int * nodes_v1, * adjacency_list_v1, * nodes_transpose_v1, * adjacency_list_transpose_v1;
	bool * is_u;
	
	num_nodes_v1 = (int)num_nodes;
	num_edges_v1 = (int)num_edges;
	nodes_v1 = (int *) malloc((num_nodes_v1+1) * sizeof(int));
	adjacency_list_v1 = (int *) malloc(num_edges_v1 * sizeof(int));
	nodes_transpose_v1 = (int *) malloc((num_nodes_v1+1) * sizeof(int));
	adjacency_list_transpose_v1 = (int *) malloc(num_edges_v1 * sizeof(int));
	is_u = (bool *) malloc(num_nodes_v1 * sizeof(bool));

	for(int i=0; i<num_nodes; i++){
		nodes_v1[i] = (int)nodes[i];
		nodes_transpose_v1[i] = (int)nodes_transpose[i];
		is_u[i] = get_is_u(og_status[i]);
	}

	for(int i=0; i<num_edges; i++){
		adjacency_list_v1[i] = (int)adjacency_list[i];
		adjacency_list_transpose_v1[i] = (int)adjacency_list_transpose[i];
	}

	char * status;
	status = (char *) malloc(num_nodes * sizeof(char));

	vector<double> executionTimes;
	/* printf("Versione 0 -Sequen.-\n");
	for(int i=0;i<repeat + WARMUP;i++){
		memcpy(status, og_status, num_nodes);
		
		auto start = chrono::high_resolution_clock::now();
		routine(num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status);
		auto end = chrono::high_resolution_clock::now();
		if (WARMUP < i) {
			executionTimes.push_back(chrono::duration<double, milli>(end - start).count());
		}
	}

  	print_benchmark(executionTimes); 
	executionTimes.clear(); */

	printf("Versione 1 -Naive-\n");
	for(int i=0;i<repeat + WARMUP;i++){
		auto start = chrono::high_resolution_clock::now();
		routine_v1(num_nodes_v1, num_edges_v1, nodes_v1, adjacency_list_v1, nodes_transpose_v1, adjacency_list_transpose_v1, is_u);
		auto end = chrono::high_resolution_clock::now();

		if (WARMUP < i) {
			executionTimes.push_back(chrono::duration<double, milli>(end - start).count());
		}
		
		hipDeviceReset();
	}

	print_benchmark(executionTimes);
	executionTimes.clear();

 	printf("Versione 2 -Status-\n");
	print_benchmark(common_routine(routine_v2, num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status, og_status, repeat));
	
 	printf("Versione 3 -Streams-\n");
	print_benchmark(common_routine(routine_v3, num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status, og_status, repeat));
	
	printf("Versione 4 -Pinned-\n");
	print_benchmark(common_routine(routine_v4, num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status, og_status, repeat));
	
 	printf("Versione 5 -OpenMP-\n");
	print_benchmark(common_routine(routine_v5, num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status, og_status, repeat));
	
	printf("Versione 6 -Reach Opt.-\n");
	print_benchmark(common_routine(routine_v6, num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status, og_status, repeat));
 
	printf("Versione 7 -Status unico-\n");
	print_benchmark(common_routine(routine_v7, num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, status, og_status, repeat));
} 