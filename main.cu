#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include "adj_list.cpp"
#include "reverse_adj_list.cpp"

#define debug 1
using namespace std;

__global__ void trimming(int * nodes, int * nodes_transpose, int * adjacency_list_transpose, int num_nodes, int num_edges, bool * forward_visited, bool * backward_visited, int * subgraph, bool * forward_terminate) {
	int i, list_pointer_1, list_pointer_2;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	bool elim = true;

	if (id < num_nodes) {
		printf("FW :: v[%d]=%d : v[%d]=%d \n", id, nodes[id], id + 1, nodes[id + 1]);
		//printf("BW :: v[%d]=%d : v[%d]=%d \n", id, nodes_transpose[id], id+1, nodes_transpose[id+1]);

		if (forward_visited[id] == false) {

			if (id == num_nodes - 1) {
				elim = true;
				if (nodes[id] == num_edges || nodes_transpose[id] == num_edges) {
					forward_visited[id] = true;
					backward_visited[id] = true;
					subgraph[id] = 4 * id + 1;
					* forward_terminate = false;
					printf("Trim e : v[%d] : %d, setting forward_terminate to %d \n", id, subgraph[id], * forward_terminate);
				}
			} else if ((nodes[id] == nodes[id + 1]) || (nodes_transpose[id] == nodes_transpose[id + 1])) {
				forward_visited[id] = true;
				backward_visited[id] = true;
				subgraph[id] = 4 * id + 1;
				* forward_terminate = false;
				printf("Trim e : v[%d] : %d, setting forward_terminate to %d \n", id, subgraph[id], * forward_terminate);
			} else {
				list_pointer_1 = nodes_transpose[id];

				if (id == num_nodes - 1)
					list_pointer_2 = num_edges;
				else
					list_pointer_2 = nodes_transpose[id + 1];

				for (i = list_pointer_1; i < list_pointer_2; i++) {
					printf("iteration %d subgraph[%d]=%d, subgraph[%d]=%d\n", i, adjacency_list_transpose[i] - 1, subgraph[adjacency_list_transpose[i] - 1], id, subgraph[id]);
					if (subgraph[adjacency_list_transpose[i] - 1] == subgraph[id]) {
						elim = false;
						break;
					}
				}
				if (elim == true) {
					forward_visited[id] = true;
					backward_visited[id] = true;
					subgraph[id] = 4 * id + 1;
					* forward_terminate = false;
					printf("Trim e : v[%d] : %d, setting forward_terminate to %d,subgraph to %d \n", id, subgraph[id], * forward_terminate, subgraph[id]);
				}

			}
		}
	}
}

__global__ void forward_closure(int * device_nodes, int * device_adjacency_list, int * subgraph, bool * forward_visited, bool * forward_terminate, int num_nodes, int num_edges) {
	//	printf("in fw\n");
	int i, list_pointer_1, list_pointer_2;

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pivot = id + 1;
	if (id < num_nodes) {
		//		printf("TID = %d v : %d sg %d\n",pivot, forward_visited[id], subgraph[id]);

		if (forward_visited[id]) {
			list_pointer_1 = device_nodes[pivot - 1];

			if (pivot == num_nodes)
				list_pointer_2 = num_edges;
			else
				list_pointer_2 = device_nodes[pivot];

			//			printf("id = %d :: %d %d \n", id, list_pointer_1, list_pointer_2);	
			for (i = list_pointer_1; i < list_pointer_2; i++) {
				//				printf("v[%d] : %d sp=%d s=%d \n", device_adjacency_list[i], forward_visited[device_adjacency_list[i]-1],subgraph[pivot-1],subgraph[device_adjacency_list[i]-1]);	
				if (forward_visited[device_adjacency_list[i] - 1] == false && subgraph[pivot - 1] == subgraph[device_adjacency_list[i] - 1]) {
					//					printf("src -> dest : %d -> %d\n",pivot, device_adjacency_list[i]);
					forward_visited[device_adjacency_list[i] - 1] = true;
					* forward_terminate = false;
				}
			}
		}
	}
}

__global__ void generate_subgraph(int pivot, bool * forward_visited, bool * backward_visited, int * subgraph, int num_nodes) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < num_nodes) {
		// Il nodo è stato visitato sia dalla backward che dalla forward
		// Il nodo fa parte di una SCC
		if (forward_visited[id] == backward_visited[id] && forward_visited[id] == true) {
			subgraph[id] = 4 * pivot;
		}

		// Il nodo "id" è stato visitato dalla forward, ma non dalla backward
		// Si deve calcolare FB(F\B)
		if (forward_visited[id] != backward_visited[id] && forward_visited[id] == true) {
			subgraph[id] = 4 * pivot + 1;
			forward_visited[id] = backward_visited[id] = false;
		}

		// Il nodo "id" è stato visitato dalla backward, ma non dalla forward
		// Si deve calcolare FB(B\F)
		if (forward_visited[id] != backward_visited[id] && backward_visited[id] == true) {
			subgraph[id] = 4 * pivot + 2;
			forward_visited[id] = backward_visited[id] = false;
		}

		// Il nodo non è stato visitato da nessuno
		// Si deve calcolare FB( V \ (B U F))
		if (forward_visited[id] == backward_visited[id] && backward_visited[id] == false) {
			subgraph[id] = 4 * pivot + 3;
			forward_visited[id] = backward_visited[id] = false;
		}
	}
}

void fw_bw(int num_nodes, int num_edges, int * nodes, int * adjacency_list, int * nodes_transpose, int * adjacency_list_transpose) {
	int * device_nodes, * device_adjacency_list, * device_nodes_transpose, * device_adjacency_list_transpose;
	int * subgraph;
	bool * forward_visited, * backward_visited;
	bool * forward_terminate, * backward_terminate, * device_forward_terminate, * device_backward_terminate;
	int i = 0;

	hipMalloc((void ** ) & device_nodes, num_nodes * (sizeof(int)));
	hipMalloc((void ** ) & device_adjacency_list, num_edges * (sizeof(int)));
	hipMalloc((void ** ) & device_nodes_transpose, num_nodes * (sizeof(int)));
	hipMalloc((void ** ) & device_adjacency_list_transpose, num_edges * (sizeof(int)));
	hipMalloc((void ** ) & subgraph, num_nodes * (sizeof(int)));
	hipMalloc((void ** ) & forward_visited, num_nodes * (sizeof(bool)));
	hipMalloc((void ** ) & backward_visited, num_nodes * (sizeof(bool)));

	hipHostAlloc((void ** ) & forward_terminate, 1 * sizeof(bool), hipHostMallocMapped);
	hipHostAlloc((void ** ) & backward_terminate, 1 * sizeof(bool), hipHostMallocMapped);

	hipMemset(subgraph, 0, num_nodes * sizeof(int));
	hipMemset(forward_visited, false, num_nodes);
	hipMemset(backward_visited, false, num_nodes);

	hipHostGetDevicePointer( & device_forward_terminate, forward_terminate, 0);
	hipHostGetDevicePointer( & device_backward_terminate, backward_terminate, 0);

	hipMemcpy(device_nodes, nodes, sizeof(int) * num_nodes, hipMemcpyHostToDevice);
	hipMemcpy(device_adjacency_list, adjacency_list, sizeof(int) * num_edges, hipMemcpyHostToDevice);
	hipMemcpy(device_nodes_transpose, nodes_transpose, sizeof(int) * num_nodes, hipMemcpyHostToDevice);
	hipMemcpy(device_adjacency_list_transpose, adjacency_list_transpose, sizeof(int) * num_edges, hipMemcpyHostToDevice);

	int num_blocks, num_threads_per_block, pivot;
	num_threads_per_block = 256;
	num_blocks = num_nodes / num_threads_per_block + (num_nodes % num_threads_per_block == 0 ? 0 : 1);

	if (debug)
		cout << "N° blocks: " << num_blocks << " ,n° threads: " << num_threads_per_block << endl;

	// Complete Trimming
	// while (*forward_terminate == false || i < 5) {                  // Il *forward_terminate == false era commentato, ho commentato tutta la riga invece
    // DOMANDA: perchè fa 'trimming 5 volte?'
	while (i < 5) {
		* forward_terminate = true;
		trimming << < num_blocks, num_threads_per_block >>> (device_nodes, device_nodes_transpose, device_adjacency_list_transpose, num_nodes, num_edges, forward_visited, backward_visited, subgraph, device_forward_terminate);
		hipDeviceSynchronize();
		printf("terminate : %d \n", * forward_terminate);
		i++;
	}

	* forward_terminate = false;
	pivot = 0;
	hipMemset( & forward_visited[pivot], true, 1);
	hipMemset( & backward_visited[pivot], true, 1);

	//Forward-Closure
	if (debug) cout << "Forward closure\n";
	while ( * forward_terminate == false) {
		* forward_terminate = true;
		forward_closure << < num_blocks, num_threads_per_block >>> (device_nodes, device_adjacency_list, subgraph, forward_visited, device_forward_terminate, num_nodes, num_edges);
		hipDeviceSynchronize();
	}

	//Backward-Closure
	if (debug) cout << "Backward  closure\n";
	while ( * backward_terminate == false) {
		* backward_terminate = true;
		forward_closure << < num_blocks, num_threads_per_block >>> (device_nodes_transpose, device_adjacency_list_transpose, subgraph, backward_visited, device_backward_terminate, num_nodes, num_edges);
		hipDeviceSynchronize();
	}

	//Finding 4 Subgraphs		
	generate_subgraph << < num_blocks, num_threads_per_block >>> (pivot, forward_visited, backward_visited, subgraph, num_nodes);

	hipFree(device_nodes);
	hipFree(device_adjacency_list);
	hipFree(device_nodes_transpose);
	hipFree(device_adjacency_list_transpose);
	hipFree(subgraph);
	hipFree(forward_visited);
	hipFree(backward_visited);
	hipHostFree(forward_terminate);
	hipHostFree(backward_terminate);
}

int main(int argc, char ** argv) {
	if (argc != 2) {
		cout << " Invalid Usage !! Usage is ./a.out <graph_input_file> \n";
		return -1;
	}
	const char *filename = argv[1];

	//Così sembra che la prima riga sia letteralmente scartata
	std::string line;
	std::ifstream infile(filename);
	std::getline(infile, line);

	/*
	La seconda riga contiene:
	- Numero di archi
	- Numero di nodi
	- Numero bi BOH
	*/
	std::getline(infile, line);
	std::istringstream iss(line);

	char percentage_sign;
	int num_edges, num_nodes, edge_weight, i;

	iss >> percentage_sign; //Questo dovrebbe essere il simbolo "%" nei file, che viene acquisito qui, ma poi mai più usato
	iss >> num_edges;
	iss >> num_nodes;
	iss >> edge_weight;

	infile.close();

	std::cout << "Number of nodes: " << num_nodes << endl;
	std::cout << "Number of edges: " << num_edges << endl;

	//Inizializzazione delle strutture dati principali
	int *nodes = new int[num_nodes];
	int *adjacency_list = new int[num_edges];
	int *nodes_transpose = new int[num_nodes];
	int *adjacency_list_transpose = new int[num_edges];

    // Inizializzazione delle liste
	for (i = 0; i < num_nodes; i++){
		nodes[i] = 0;
		nodes_transpose[i] = 0;
	}

	// Creazione delle liste di adiacenza
	adj_list(filename, nodes, adjacency_list);

	// Creazione delle liste di adiacenza del grafo trasposto (per la backward clousure)
	// Forse si può evitare la ripetizione di codice usando il codice di adj_list leggermente modificato
	reverse_adj_list(filename, nodes_transpose, adjacency_list_transpose);

	if (debug) {
		cout << " Adj List " << endl;
		cout << " ---O(V) \n";
		for (i = 0; i < num_nodes; i++) {
			cout << "nodes[" << i << "] : " << nodes[i] << endl;
		}
		cout << " ---O(E) \n";
		for (i = 0; i < num_edges; i++) {
			cout << "adjacency_list[" << i << "] : " << adjacency_list[i] << endl;
		}

		cout << " Transpose Adj List " << endl;
		cout << " ---O(V) \n";
		for (i = 0; i < num_nodes; i++) {
			cout << "nodes[" << i << "] : " << nodes_transpose[i] << endl;
		}
		cout << " ---O(E) \n";
		for (i = 0; i < num_edges; i++) {
			cout << "adjacency_list[" << i << "] : " << adjacency_list_transpose[i] << endl;
		}
	}

	fw_bw(num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose);

	delete(nodes);
	delete(adjacency_list);
	delete(nodes_transpose);
	delete(adjacency_list_transpose);
}