#include "hip/hip_runtime.h"
#include <sstream>
#include <fstream>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#include "adj_list.cpp"
#include "reverse_adj_list.cpp"

#define debug 1
using namespace std;

// cioè almeno sembra, ma se divide in 4 sta facendo OBF? //cristo sandro

__global__ void trimming(int * Vertices, int * VerticesT, int * AdjListT, int n, int m, bool * VisitedF, bool * VisitedB, int * subgraph, bool * terminateF) {
	int i, ListPointer1, ListPointer2;
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	bool elim = true;

	if (id < n) {
		printf("FW :: v[%d]=%d : v[%d]=%d \n", id, Vertices[id], id + 1, Vertices[id + 1]);
		//printf("BW :: v[%d]=%d : v[%d]=%d \n", id, VerticesT[id], id+1, VerticesT[id+1]);

		if (VisitedF[id] == false) {

			if (id == n - 1) {
				elim = true;
				if (Vertices[id] == m || VerticesT[id] == m) {
					VisitedF[id] = true;
					VisitedB[id] = true;
					subgraph[id] = 4 * id + 1;
					* terminateF = false;
					printf("Trim e : v[%d] : %d, setting terminateF to %d \n", id, subgraph[id], * terminateF);
				}
			} else if ((Vertices[id] == Vertices[id + 1]) || (VerticesT[id] == VerticesT[id + 1])) {
				VisitedF[id] = true;
				VisitedB[id] = true;
				subgraph[id] = 4 * id + 1;
				* terminateF = false;
				printf("Trim e : v[%d] : %d, setting terminateF to %d \n", id, subgraph[id], * terminateF);
			} else {
				ListPointer1 = VerticesT[id];

				if (id == n - 1)
					ListPointer2 = m;
				else
					ListPointer2 = VerticesT[id + 1];

				for (i = ListPointer1; i < ListPointer2; i++) {
					printf("iteration %d subgraph[%d]=%d, subgraph[%d]=%d\n", i, AdjListT[i] - 1, subgraph[AdjListT[i] - 1], id, subgraph[id]);
					if (subgraph[AdjListT[i] - 1] == subgraph[id]) {
						elim = false;
						break;
					}
				}
				if (elim == true) {
					VisitedF[id] = true;
					VisitedB[id] = true;
					subgraph[id] = 4 * id + 1;
					* terminateF = false;
					printf("Trim e : v[%d] : %d, setting terminateF to %d,subgraph to %d \n", id, subgraph[id], * terminateF, subgraph[id]);
				}

			}
		}
	}
}

__global__ void forward_closure(int * dVertices, int * dAdjList, int * subgraph, bool * visitedF, bool * terminateF, int numVertices, int numEdges) {
	//	printf("in fw\n");
	int i, ListPointer1, ListPointer2;

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	int pivot = id + 1;
	if (id < numVertices) {
		//		printf("TID = %d v : %d sg %d\n",pivot, visitedF[id], subgraph[id]);

		if (visitedF[id]) {
			ListPointer1 = dVertices[pivot - 1];

			if (pivot == numVertices)
				ListPointer2 = numEdges;
			else
				ListPointer2 = dVertices[pivot];

			//			printf("id = %d :: %d %d \n", id, ListPointer1, ListPointer2);	
			for (i = ListPointer1; i < ListPointer2; i++) {
				//				printf("v[%d] : %d sp=%d s=%d \n", dAdjList[i], visitedF[dAdjList[i]-1],subgraph[pivot-1],subgraph[dAdjList[i]-1]);	
				if (visitedF[dAdjList[i] - 1] == false && subgraph[pivot - 1] == subgraph[dAdjList[i] - 1]) {
					//					printf("src -> dest : %d -> %d\n",pivot, dAdjList[i]);
					visitedF[dAdjList[i] - 1] = true;
					* terminateF = false;
				}
			}
		}
	}
}

__global__ void generate_subgraph(int pivot, bool * visitedF, bool * visitedB, int * subgraph, int numVertices) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < numVertices) {
		// Il nodo è stato visitato sia dalla backward che dalla forward
		// Il nodo fa parte di una SCC
		if (visitedF[id] == visitedB[id] && visitedF[id] == true) {
			subgraph[id] = 4 * pivot;
		}

		// Il nodo "id" è stato visitato dalla forward, ma non dalla backward
		// Si deve calcolare FB(F\B)
		if (visitedF[id] != visitedB[id] && visitedF[id] == true) {
			subgraph[id] = 4 * pivot + 1;
			visitedF[id] = visitedB[id] = false;
		}

		// Il nodo "id" è stato visitato dalla backward, ma non dalla forward
		// Si deve calcolare FB(B\F)
		if (visitedF[id] != visitedB[id] && visitedB[id] == true) {
			subgraph[id] = 4 * pivot + 2;
			visitedF[id] = visitedB[id] = false;
		}

		// Il nodo non è stato visitato da nessuno
		// Si deve calcolare FB( V \ (B U F))
		if (visitedF[id] == visitedB[id] && visitedB[id] == false) {
			subgraph[id] = 4 * pivot + 3;
			visitedF[id] = visitedB[id] = false;
		}
	}
}

void fw_bw(int n, int m, int * Vertices, int * AdjacencyList, int * Vertices_Transpose, int * AdjacencyList_Transpose) {
	int * dVertices, * dAdjList, * dVerticesT, * dAdjListT;
	int * subgraph;
	bool * visitedF, * visitedB;
	bool * terminateF, * terminateB, * dterminateF, * dterminateB;
	int i = 0;

	hipMalloc((void ** ) & dVertices, n * (sizeof(int)));
	hipMalloc((void ** ) & dAdjList, m * (sizeof(int)));
	hipMalloc((void ** ) & dVerticesT, n * (sizeof(int)));
	hipMalloc((void ** ) & dAdjListT, m * (sizeof(int)));
	hipMalloc((void ** ) & subgraph, n * (sizeof(int)));
	hipMalloc((void ** ) & visitedF, n * (sizeof(bool)));
	hipMalloc((void ** ) & visitedB, n * (sizeof(bool)));

	hipHostAlloc((void ** ) & terminateF, 1 * sizeof(bool), hipHostMallocMapped);
	hipHostAlloc((void ** ) & terminateB, 1 * sizeof(bool), hipHostMallocMapped);

	hipMemset(subgraph, 0, n * sizeof(int));
	hipMemset(visitedF, false, n);
	hipMemset(visitedB, false, n);

	hipHostGetDevicePointer( & dterminateF, terminateF, 0);
	hipHostGetDevicePointer( & dterminateB, terminateB, 0);

	hipMemcpy(dVertices, Vertices, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(dAdjList, AdjacencyList, sizeof(int) * m, hipMemcpyHostToDevice);
	hipMemcpy(dVerticesT, Vertices_Transpose, sizeof(int) * n, hipMemcpyHostToDevice);
	hipMemcpy(dAdjListT, AdjacencyList_Transpose, sizeof(int) * m, hipMemcpyHostToDevice);

	int numBlocks, numThreadsPerBlock, pivot;
	numThreadsPerBlock = 256;
	numBlocks = n / numThreadsPerBlock + (n % numThreadsPerBlock == 0 ? 0 : 1);

	if (debug)
		cout << "N° blocks: " << numBlocks << " ,n° threads: " << numThreadsPerBlock << endl;

	// Complete Trimming
	FWD reach c'è (anche bwd), trimming c'è pivot non ho visto come sia
	// Quindi prima di tutto chiama trimming, yess
	while ( /**terminateF == false ||*/ i < 5) {
		* terminateF = true;
		trimming << < numBlocks, numThreadsPerBlock >>> (dVertices, dVerticesT, dAdjListT, n, m, visitedF, visitedB, subgraph, dterminateF);
		hipDeviceSynchronize();
		printf("terminate : %d \n", * terminateF);
		i++;
	}

	* terminateF = false;
	pivot = 0; //bella
	hipMemset( & visitedF[pivot], true, 1);
	hipMemset( & visitedB[pivot], true, 1);

	//Forward-Closure
	if (debug) cout << "Forward closure\n";
	while ( * terminateF == false) {
		* terminateF = true;
		forward_closure << < numBlocks, numThreadsPerBlock >>> (dVertices, dAdjList, subgraph, visitedF, dterminateF, n, m);
		hipDeviceSynchronize();
	}

	//Backward-Closure
	if (debug) cout << "Backward  closure\n";
	while ( * terminateB == false) {
		* terminateB = true;
		forward_closure << < numBlocks, numThreadsPerBlock >>> (dVerticesT, dAdjListT, subgraph, visitedB, dterminateB, n, m);
		hipDeviceSynchronize();
	}

	//Finding 4 Subgraphs		
	generate_subgraph << < numBlocks, numThreadsPerBlock >>> (pivot, visitedF, visitedB, subgraph, n);

	// Tu ti ricordi come si fa la ricorsione in cuda? io no
	// forse va fatto con la programmazione dinamica?
	hipFree(dVertices);
	hipFree(dAdjList);
	hipFree(dVerticesT);
	hipFree(dAdjListT);
	hipFree(subgraph);
	hipFree(visitedF);
	hipFree(visitedB);
	hipHostFree(terminateF);
	hipHostFree(terminateB);
}

int main(int argc, char ** argv) {
	if (argc != 2) {
		cout << " Invalid Usage !! Usage is ./a.out <graph_input_file> \n";
		return -1;
	}
	const char *filename = argv[1];

	//Così sembra che la prima riga sia letteralmente scartata
	std::string line;
	std::ifstream infile(filename);
	std::getline(infile, line);

	/*
	La seconda riga contiene:
	- Numero di archi
	- Numero di nodi
	- Numero bi BOH
	*/
	std::getline(infile, line);
	std::istringstream iss(line);

	char c;
	int m, n, x, i;

	iss >> c; //Questo dovrebbe essere il simbolo "%" nei file, che viene acquisito qui, ma poi mai più usato
	iss >> m;
	iss >> n;
	iss >> x;

	std::cout << "Number of vertices: " << n << endl;
	std::cout << "Number of edges: " << m << endl;

	//Inizializzazione delle strutture dati principali
	int *Vertices = new int[n];
	int *AdjacencyList = new int[m];

	int *Vertices_Transpose = new int[n];
	int *AdjacencyList_Transpose = new int[m];

	for (i = 0; i < n; i++){
		Vertices[i] = 0;
		Vertices_Transpose[i] = 0;
	}

	infile.close();

	// Creazione delle liste di adiacenza
	adj_list(filename, Vertices, AdjacencyList);

	// Creazione delle liste di adiacenza del grafo trasposto (per la backward clousure)
	// Forse si può evitare la ripetizione di codice usando il codice di adj_list leggermente modificato
	reverse_adj_list(filename, Vertices_Transpose, AdjacencyList_Transpose);

	if (debug) {
		cout << " Adj List " << endl;
		cout << " ---O(V) \n";
		for (i = 0; i < n; i++) {
			cout << "Vertices[" << i << "] : " << Vertices[i] << endl;
		}
		cout << " ---O(E) \n";
		for (i = 0; i < m; i++) {
			cout << "AdjacencyList[" << i << "] : " << AdjacencyList[i] << endl;
		}

		cout << " Transpose Adj List " << endl;
		cout << " ---O(V) \n";
		for (i = 0; i < n; i++) {
			cout << "Vertices[" << i << "] : " << Vertices_Transpose[i] << endl;
		}
		cout << " ---O(E) \n";
		for (i = 0; i < m; i++) {
			cout << "AdjacencyList[" << i << "] : " << AdjacencyList_Transpose[i] << endl;
		}
	}

	fw_bw(n, m, Vertices, AdjacencyList, Vertices_Transpose, AdjacencyList_Transpose);

	delete(Vertices);
	delete(AdjacencyList);
	delete(Vertices_Transpose);
	delete(AdjacencyList_Transpose);
}