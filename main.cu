#include "hip/hip_runtime.h"
#include "utils.cpp"
#include <cstring>
#include <hip/hip_runtime.h>
using namespace std;

#define DEBUG_F_KERNEL false
#define DEBUG_REACH false
#define DEBUG_TRIMMING_KERNEL false
#define DEBUG_TRIMMING false
#define DEBUG_UPDATE false
#define DEBUG_FW_BW false
#define DEBUG_MAIN false
#define DEBUG_FINAL true

static void handle_error(hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
		exit( EXIT_FAILURE );
	}
}
#define HANDLE_ERROR( err ) (handle_error( err, __FILE__, __LINE__ ))

__global__ void f_kernel(int num_nodes, int num_edges, int * d_nodes, int * d_adjacency_list, int * d_pivots, bool * d_is_visited, bool * d_is_eliminated, bool * d_is_expanded, bool * d_stop){
	// Esecuzione di un thread della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

    // Per ogni nodo
	if(v < num_nodes) {

        // Si controlla se non è stato eliminato E è stato eliminato E non è stato espanso
		if(!d_is_eliminated[v] && d_is_visited[v] && !d_is_expanded[v]) {
            // Si segna come espanso
			d_is_expanded[v] = true;

            // Per ogni nodo a cui punta
			for(int u = d_nodes[v]; u < d_nodes[v + 1]; u++) {	
				int dst = d_adjacency_list[u];

                // Si controlla se non è stato eliminato E se non è stato visitato E se il colore del nodo che punta corrisponde a quello del nodo puntato
				if(!d_is_eliminated[dst] && !d_is_visited[dst] && d_pivots[v] == d_pivots[dst]) {
                    // Setta il nodo puntato a visitato
					d_is_visited[dst] = true;
                    // Permette di continuare il ciclo in reach, perchè si è trovato un altro nodo da visitare
					*d_stop = false;
				}
			}
		}
	}
}

void reach(int num_nodes, int num_edges, int * d_nodes, int * d_adjacency_list, int * d_pivots, bool * d_is_visited, bool * d_is_eliminated, bool * d_is_expanded, const int n_blocks, const int t_per_blocks) {
	// Esecuzione ricorsiva della chiusura in avanti/indietro
	// @param:	pivots			=	Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno
	// 			is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return 	is_visited		=	Lista che per ogni 'v' dice se è stato visitato dalla reach o meno, aggiornata dopo l'esecuzione del trimming
	// 			is_expanded		=	Lista che per ogni 'v' dice se sono stato visitati i figli diretti o meno, aggiornata dopo l'esecuzione del trimming

	bool stop, *d_stop;
	stop = false;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));
	
    // Si effettua la chiusura in avanti/indietro
    while(!stop) {
		HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
        f_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, num_edges, d_nodes, d_adjacency_list, d_pivots, d_is_visited, d_is_eliminated, d_is_expanded, d_stop);	
		HANDLE_ERROR(hipMemcpy(&stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
    }
	
	HANDLE_ERROR(hipFree(d_stop));
}

__global__ void trimming_kernel(int num_nodes, int * d_nodes, int * d_nodes_transpose, int * d_adjacency_list,  int * d_adjacency_list_transpose, bool * d_is_eliminated, bool * d_stop){
	// Esegue un'eliminazione di nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes) {
		if(!d_is_eliminated[v]){
			// Se questo valore non verrà cambiato, allora il nodo verrà cancellato
			bool elim = true;

			bool forward = false;
			bool backward = false;
			
			// Nel caso un nodo abbia entrambi in_degree o out_degree diversi da 0, tra i soli nodi non eliminati, allora non va eliminato
			for(int u = d_nodes[v]; u < d_nodes[v+1]; u++){
				if(!d_is_eliminated[d_adjacency_list[u]]) {
					forward = true;
				}
			}
			if(forward) {
				for(int u = d_nodes_transpose[v]; u < d_nodes_transpose[v+1]; u++){
					if(!d_is_eliminated[d_adjacency_list_transpose[u]]) {
						backward = true;
					}
				}
			}
			if(backward) {
				elim = false;
			}

			if(elim){
				d_is_eliminated[v] = true;
				*d_stop = false;
			}
		}
	}
}

void trimming(int num_nodes, int num_edges, int * d_nodes, int * d_nodes_transpose, int * d_adjacency_list, int * d_adjacency_list_transpose, bool * d_is_eliminated, const int n_blocks, const int t_per_blocks) {
	// Elimina iterativamente i nodi con out-degree o in-degree uguale a 0, senza contare i nodi eliminati
	// @param:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// @return:	is_eliminated	=	Lista che per ogni 'v' dice se il nodo è stato eliminato o no, aggiornata dopo l'esecuzione del trimming

	bool stop, *d_stop;
	stop = false;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));

    while(!stop) {
		HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
        trimming_kernel<<<n_blocks, t_per_blocks>>>(num_nodes, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_is_eliminated, d_stop);
		HANDLE_ERROR(hipMemcpy(&stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
    }

	HANDLE_ERROR(hipFree(d_stop));
}

__global__ void set_colors(int num_nodes, bool * d_fw_is_visited, bool * d_bw_is_visited, int * d_pivots, int * d_colors, bool * d_is_eliminated, long * d_write_id_for_pivots, bool * d_stop){
	// Esegue l'update dei valori del pivot facendo una race, scrivendo il "colore" di una serie di pivot in array simultaneamente
	// @param:	pivots						= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated				= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited				= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited				= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: d_write_id_for_pivots		= Lista che conterrà, nelle posizione identificate dai colori appena calcolati, i nuovi pivot da assegnare
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes) {
		if(d_is_eliminated[v]){
			d_pivots[v] = v;
		} 
		
		if(d_fw_is_visited[v] == d_bw_is_visited[v] && d_fw_is_visited[v] == true){
			d_colors[v] = 4 * d_pivots[v];
		} else {
			if(d_fw_is_visited[v] != d_bw_is_visited[v] && d_fw_is_visited[v] == true){
				d_colors[v] = 4 * d_pivots[v] + 1;
			}else if(d_fw_is_visited[v] != d_bw_is_visited[v] && d_fw_is_visited[v] == false){
				d_colors[v] = 4 * d_pivots[v] + 2;
			}else if(d_fw_is_visited[v] == d_bw_is_visited[v] && d_fw_is_visited[v] == false){
				d_colors[v] = 4 * d_pivots[v] + 3;				
			}
				
			if(!d_is_eliminated[v]){
				*d_stop = false;
			}
		}
		d_write_id_for_pivots[d_colors[v]] = v;
	}
}

__global__ void set_race_winners(int num_nodes, bool * d_is_eliminated, int * d_pivots, int * d_colors, long * d_write_id_for_pivots, bool * d_fw_is_visited, bool * d_bw_is_visited){
	// Ottenuti i valori della race, si vanno ad impostare i nuovi pivot
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update

	
	int v = threadIdx.x + blockIdx.x * blockDim.x;
	if(v < num_nodes) {
		// Se il nodo è stato eliminato, allora il suo pivot è per forza se stesso
		if(d_is_eliminated[v]){
			d_pivots[v] = v;
		}else{
			d_pivots[v] = d_write_id_for_pivots[d_colors[v]];
			d_fw_is_visited[d_pivots[v]] = true;
			d_bw_is_visited[d_pivots[v]] = true;
		}
	}
}

__global__ void initialize_pivot(int num_nodes, bool * d_is_eliminated, int * d_pivots, bool * d_fw_is_visited, bool * d_bw_is_visited) {
	// Scelta iniziale del primo pivot, basandosi sui nodi cancellati inizialmente
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, avente come pivot un nodo non cancellato
	//          fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no. A questo punto l'unico nodo visitato è il solo pivot scelto
	//          bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no. A questo punto l'unico nodo visitato è il solo pivot scelto

	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes){
		__shared__ int chosen_pivot;
		if(!d_is_eliminated[v]){
			chosen_pivot = v;
		}

		// Sincronizziamo qui i thread per inizializzare questi array: lanciare un altro thread
		// solo per inizializzare gli array potrebbe risultare più pesante che farlo qui
		__syncthreads();

		d_pivots[v] = chosen_pivot;
		d_fw_is_visited[d_pivots[v]] = true;
		d_bw_is_visited[d_pivots[v]] = true;
	}
}

void update(int num_nodes, int * d_pivots, bool * d_fw_is_visited, bool * d_bw_is_visited, bool * d_is_eliminated, long * d_write_id_for_pivots, int * d_colors, bool * stop, const int n_blocks, const int t_per_blocks) {
	// Esegue l'update dei valori del pivot facendo una race
	// @param:	pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene
	// 			is_eliminated	= Lista che per ogni 'v' dice se il nodo è stato eliminato o no
	// 			fw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la forward reach partendo dai pivots o no
	// 			bw_is_visited	= Lista che per ogni 'v' dice se il nodo è stato visitato con la backward reach partendo dai pivots o no
	// @return: pivots			= Lista che contiene, per ogni 'v', il valore del pivot della SCC a cui tale nodo 'v' appartiene, aggiornata dopo l'esecuzione di update


	bool *d_stop;

	HANDLE_ERROR(hipMalloc((void**)&d_stop, sizeof(bool)));
	
	HANDLE_ERROR(hipMemset(d_write_id_for_pivots, -1, 4 * num_nodes * sizeof(long)));
	HANDLE_ERROR(hipMemset(d_stop, true, sizeof(bool)));
	
	// Dai paper:
	// These subgraphs are 
	// 		1) the strongly connected component with the pivot;
	// 		2) the subgraph given by vertices in the forward closure but not in the backward closure; 
	// 		3) the subgraph given by vertices in the backward closure but not in the forward closure;
	// 		4) the subgraph given by vertices that are neither in the forward nor in the backward closure.
	
	// The subgraphs that do not contain the pivot form three independent instances of the same problem, and therefore, 
	// they are recursively processed in parallel with the same algorithm
	
	set_colors<<<n_blocks, t_per_blocks>>>(num_nodes, d_fw_is_visited, d_bw_is_visited, d_pivots, d_colors, d_is_eliminated, d_write_id_for_pivots, d_stop);
	
	HANDLE_ERROR(hipMemcpy(stop, d_stop, sizeof(bool), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(d_stop));

	// Setto i valori dei pivot che hanno vinto la race
	// Se sono stati eliminati, allora setta il valore dello stesso nodo 
	set_race_winners<<<n_blocks, t_per_blocks>>>(num_nodes, d_is_eliminated, d_pivots, d_colors, d_write_id_for_pivots, d_fw_is_visited, d_bw_is_visited);
}

__global__ void trim_u_kernel(int num_nodes, int * d_nodes, int * d_adjacency_list, int * d_pivots, bool * d_is_u, int * d_is_scc){
	// Setta i pivot delle SCC uguale a -1 se questi ricevono archi da nodi u
	// param: 	pivots = 	Lista che per ogni 'v' dice il valore del pivot della SCC
	// 			is_scc =	Lista copia di pivots
	// @return:	is_scc =	Lista contenente i pivot delle SCC, però i pivot delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1
	
	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if(v < num_nodes){
		if(d_is_u[v]){
			for(int u = d_nodes[v]; u < d_nodes[v+1]; ++u) {
				if(d_pivots[v] != d_pivots[d_adjacency_list[u]]) {
					d_is_scc[d_pivots[d_adjacency_list[u]]] = -1;
				}
			}
		}
	}

}

__global__ void trim_u_propagation(int num_nodes, int * d_pivots, int * d_is_scc) {
	// Se alcuni pivot sono settati a -1, per la cancellazione dovuta a collegamenti con nodi u, 
	// propaga la cancellazione agli altri membri della SCC
	// param: 	pivots = 	Lista contenente i pivot delle SCC
	// 			is_scc =	Lista contenente i pivot delle SCC, però i pivot delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1
	// @return:	is_scc =	Lista contenente i pivot delle SCC, però i pivot e gli altri nodi delle SCC 
	// 						che ricevono archi da nodi u sono settati a -1

	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if (v < num_nodes)
		d_is_scc[v] = d_is_scc[d_pivots[v]];
}

__global__ void calculate_more_than_one(int num_nodes, int * d_more_than_one_dev, int * is_scc_dev) {
	// Trova il numero di elementi nella SCC
	// @param: is_scc =	Lista contenente i pivot delle SCC, però i pivot e gli altri nodi delle SCC 
	// 					che ricevono archi da nodi u sono settati a -1
	// @return:	more_than_one = 	Lista che per ogni nodo 'v' dice se questo è un pivot.
	// 								Se 'v' è pivot: 	more_than_one[v] = numero di elementi nella sua SCC,
	// 								Se 'v' non è pivot:	more_than_one[v] = 1

	int u = threadIdx.x + blockIdx.x * blockDim.x;

	if (u < num_nodes){
		if(is_scc_dev[u] != -1){
			// atomicAdd può essere migliorato -> Simile al problema dell'istogramma
			atomicAdd(&d_more_than_one_dev[is_scc_dev[u]], 1);
		}
	}
}

__global__ void is_scc_adjust(int num_nodes, int * more_than_one_dev, int * is_scc_dev) {
	// Restituisce una lista che dice se il nodo 'v' fa parte di una SCC
	// @param: more_than_one = 	Lista che per ogni nodo 'v' dice se questo è un pivot.
	// 							Se 'v' è pivot: 								more_than_one[v] = numero di elementi nella sua SCC,
	// 							Se 'v' non è pivot, ma fa parte di una SCC:		more_than_one[v] = 0
	// 							Se 'v' non è pivot e non fa parte di una SCC:	more_than_one[v] = 0
	// @return: is_scc =	Lista che per ogni nodo 'v' dice se questo fa parte di una SCC.
	// 						Se fa parte di una SCC: 	is_scc[v] = valore del pivot,
	// 						Se non fa parte di una SCC:	is_scc[v] = -1

	int v = threadIdx.x + blockIdx.x * blockDim.x;

	if (v < num_nodes){
		if(more_than_one_dev[v] == 1)
			is_scc_dev[v] = -1;
	}
}

int count_distinct_scc(int is_scc[], int num_nodes){
	// Restituisce il numero di SCC valide presenti nell'array is_scc
	// Questa funzione non viene parallelizzata poiché utilizzata solamente per verificare la correttezza del risultato
	// @param:  is_scc 	= 	Lista contenente le SCC valide trovate
	// @return: res    	=	Numero di SCC valide diverse

    int res = 0;
 
    // Per tutti gli elementi dell'array
    for (int i = 1; i < num_nodes; i++) {
        int j = 0;
        for (j = 0; j < i; j++)
            if (is_scc[i] == is_scc[j])
                break;
 
        // Se non è già stato contato, contalo
        if (i == j)
            res++;
    }
    return res;
}

int main(int argc, char ** argv) {
	// Impostazione del device
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);

    if (argc != 2) {
		cout << " Invalid Usage !! Usage is ./main.out <graph_input_file> \n";
		return -1;
	}

	int num_nodes, num_edges;
    int * nodes, * adjacency_list, * nodes_transpose, * adjacency_list_transpose, * is_scc;
	bool * is_eliminated, * is_u;

    create_graph_from_filename(argv[1], num_nodes, num_edges, nodes, adjacency_list, nodes_transpose, adjacency_list_transpose, is_u);

	if(DEBUG_MAIN){
		for (int i = 0; i < num_nodes; i++)
			DEBUG_MSG("nodes[" + to_string(i) + "] = ", nodes[i], DEBUG_MAIN);
		for (int i = 0; i < num_edges; i++)
			DEBUG_MSG("adjacency_list[" + to_string(i) + "] = ", adjacency_list[i], DEBUG_MAIN);
		for (int i = 0; i < num_nodes; i++)
			DEBUG_MSG("nodes_transpose[" + to_string(i) + "] = ", nodes_transpose[i], DEBUG_MAIN);
		for (int i = 0; i < num_edges; i++)
			DEBUG_MSG("adjacency_list_transpose[" + to_string(i) + "] = ", adjacency_list_transpose[i], DEBUG_MAIN);
		for (int i = 0; i < num_nodes; i++)
			DEBUG_MSG("is_u[" + to_string(i) + "] = ", is_u[i], DEBUG_MAIN);
	}

	const int THREADS_PER_BLOCK = prop.maxThreadsPerBlock;
	const int NUMBER_OF_BLOCKS = num_nodes / THREADS_PER_BLOCK + (num_nodes % THREADS_PER_BLOCK == 0 ? 0 : 1);

	// Dichiarazioni di variabili device
	int * d_is_scc, * d_more_than_one, * d_nodes, * d_adjacency_list, * d_nodes_transpose, * d_adjacency_list_transpose, * d_pivots, * d_colors;
	bool * d_is_u, * d_is_eliminated, * d_fw_is_visited, * d_bw_is_visited, * d_fw_is_expanded, * d_bw_is_expanded;
	long * d_write_id_for_pivots;

	HANDLE_ERROR(hipMalloc((void**)&d_nodes, (num_nodes+1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_nodes_transpose, (num_nodes+1) * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_adjacency_list, num_edges * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_adjacency_list_transpose, num_edges * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_pivots, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_colors, num_nodes * sizeof(int)));
	
	HANDLE_ERROR(hipMalloc((void**)&d_is_eliminated, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMalloc((void**)&d_fw_is_visited, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMalloc((void**)&d_bw_is_visited, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMalloc((void**)&d_fw_is_expanded, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMalloc((void**)&d_bw_is_expanded, num_nodes * sizeof(bool)));

	HANDLE_ERROR(hipMalloc((void**)&d_write_id_for_pivots, 4 * num_nodes * sizeof(long)));

	// Le strutture principali le copiamo nel device già qui, visto che non verranno mai modificate
	HANDLE_ERROR(hipMemcpy(d_nodes, nodes, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_adjacency_list, adjacency_list, num_edges * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_nodes_transpose, nodes_transpose, (num_nodes+1) * sizeof(int), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_adjacency_list_transpose, adjacency_list_transpose, num_edges * sizeof(int), hipMemcpyHostToDevice));

	is_eliminated = (bool*) malloc(num_nodes * sizeof(bool));

	for (int i = 0; i < num_nodes; i++){
		is_eliminated[i] = !is_u[i];
	}
	
	HANDLE_ERROR(hipMemcpy(d_is_eliminated, is_eliminated, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(d_fw_is_visited, false, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMemset(d_bw_is_visited, false, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMemset(d_fw_is_expanded, false, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMemset(d_bw_is_expanded, false, num_nodes * sizeof(bool)));
	
	// Primo trimming per eliminare i nodi che, dopo la cancellazione dei nodi non in U,
	// non avevano più out-degree e in-degree diverso da 0
	trimming(num_nodes, num_edges, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_is_eliminated, THREADS_PER_BLOCK, NUMBER_OF_BLOCKS);
	
	// Si fanno competere i thread per scelgliere un nodo che farà da pivot, a patto che quest'ultimo sia non eliminato
	initialize_pivot<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_is_eliminated, d_pivots, d_fw_is_visited, d_bw_is_visited);
	
    bool stop = false;
	
	// Si ripete il ciclo fino a quando tutti i nodi vengono eliminati
    while (!stop){
		// Forward reach
		DEBUG_MSG("Forward reach:" , "", DEBUG_FW_BW);
        reach(num_nodes, num_edges, d_nodes, d_adjacency_list, d_pivots, d_fw_is_visited, d_is_eliminated, d_fw_is_expanded, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
		
		// Backward reach
        DEBUG_MSG("Backward reach:" , "", DEBUG_FW_BW);
		reach(num_nodes, num_edges, d_nodes_transpose, d_adjacency_list_transpose, d_pivots, d_bw_is_visited, d_is_eliminated, d_bw_is_expanded, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);

		// Trimming per eliminare ulteriori nodi che non hanno più out-degree e in-degree diversi da 0
		DEBUG_MSG("Trimming:" , "", DEBUG_FW_BW);
        trimming(num_nodes, num_edges, d_nodes, d_nodes_transpose, d_adjacency_list, d_adjacency_list_transpose, d_is_eliminated, THREADS_PER_BLOCK, NUMBER_OF_BLOCKS);

		// Update dei pivot
		DEBUG_MSG("Update:" , "", DEBUG_FW_BW);
		update(num_nodes, d_pivots, d_fw_is_visited, d_bw_is_visited, d_is_eliminated, d_write_id_for_pivots, d_colors, &stop, NUMBER_OF_BLOCKS, THREADS_PER_BLOCK);
    }
	
	// Tramite fw_bw_ abbiamo ottenuto, per ogni nodo, il pivot della SCC a cui appartiene.
	// Allochiamo is_scc, che alla fine avrà per ogni nodo il pivot della sua SCC se la sua SCC è accettabile, altrimenti -1
	
	// Per iniziare le assegnamo gli stessi valori di pivots, che verranno modificati in seguito
	is_scc = (int*) malloc(num_nodes * sizeof(int));
	
	// Allochiamo more_than_one, che per ogni nodo che fa da pivot viene assegnato un contatore, il quale conta quante volte appare tale pivot
	// Se appare solo una volta, allora il nodo non fa parte di nessuna SCC
	HANDLE_ERROR(hipMalloc((void**)&d_is_u, num_nodes * sizeof(bool)));
	HANDLE_ERROR(hipMalloc((void**)&d_is_scc, num_nodes * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&d_more_than_one, num_nodes * sizeof(int)));
	
	HANDLE_ERROR(hipMemcpy(d_is_scc, d_pivots, num_nodes * sizeof(int), hipMemcpyDeviceToDevice));
	HANDLE_ERROR(hipMemcpy(d_is_u, is_u, num_nodes * sizeof(bool), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemset(d_more_than_one, 0, num_nodes * sizeof(int)));

	trim_u_kernel<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_nodes, d_adjacency_list, d_pivots, d_is_u, d_is_scc);
	trim_u_propagation<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_pivots, d_is_scc);
	calculate_more_than_one<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_more_than_one, d_is_scc);
	is_scc_adjust<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(num_nodes, d_more_than_one, d_is_scc);
	
	HANDLE_ERROR(hipFree(d_pivots));
	HANDLE_ERROR(hipFree(d_is_u));
	HANDLE_ERROR(hipFree(d_is_eliminated));
	HANDLE_ERROR(hipFree(d_more_than_one));
	HANDLE_ERROR(hipFree(d_nodes));
	HANDLE_ERROR(hipFree(d_adjacency_list));
	HANDLE_ERROR(hipFree(d_nodes_transpose));
	HANDLE_ERROR(hipFree(d_adjacency_list_transpose));

	HANDLE_ERROR(hipMemcpy(is_scc, d_is_scc, num_nodes * sizeof(int), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipFree(d_is_scc));

	for (int i = 0; i < num_nodes; i++)
        DEBUG_MSG("is_scc[" + to_string(i) + "] = ", is_scc[i], false);

	DEBUG_MSG("Number of SCCs found: ", count_distinct_scc(is_scc, num_nodes), DEBUG_FINAL);
}